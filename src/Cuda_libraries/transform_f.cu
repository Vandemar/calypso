#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
//#include <sstream>

__global__
void transF_vec(int kst, int *idx_gl_1d_rlm_j, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3 grid(constants.nidx_rlm[1],1,1);
  //dim3 block(constants.nidx_rtm[0],1,1);
  int k_rtm = threadIdx.x+kst-1;
  //int j_rlm = blockIdx.x;

// 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;

  double reg0, reg1, reg2, reg3, reg4;
  double sp1, sp2, sp3; 

  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + blockIdx.x];
//  int degree = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + blockIdx.x];
  double gauss_norm = g_sph_rlm_7[blockIdx.x];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nComp = constants.ncomp;
  int istep_rtm_r = constants.istep_rtm[0];
  int istep_rtm_t = constants.istep_rtm[1];
  int istep_rtm_m = constants.istep_rtm[2];
  int istep_rlm_r = constants.istep_rlm[0];
  int istep_rlm_j = constants.istep_rlm[1];

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x] - 1;
  ip_rtm = k_rtm * constants.istep_rtm[0];
  int mdx_n = mdx_n_rlm_rtm[blockIdx.x] - 1;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm;

  int idx;
  int idx_p_rtm = blockIdx.x*nTheta; 
 
  double r_1d_rlm_r = radius_1d_rlm_r[k_rtm]; 
  int idx_sp = nComp * ( blockIdx.x*istep_rlm_j + k_rtm*istep_rlm_r); 

  for(int t=1; t<=nVector; t++) {
    sp1=sp2=sp3=0;
    for(int l_rtm=0; l_rtm<nTheta; l_rtm++) {
      ip_rtm = 3*t + nComp * (l_rtm * istep_rtm_t + mdx_p); 
      in_rtm = 3*t + nComp * (l_rtm * istep_rtm_t + mdx_n); 

      idx = idx_p_rtm + l_rtm; 
      reg0 = __dmul_rd(gauss_norm, weight_rtm[l_rtm]);
      reg1 = __dmul_rd(reg0, P_rtm[idx]);
      reg2 = __dmul_rd(reg0, dP_rtm[idx]);
      reg4 = __dmul_rd(P_rtm[idx], (double) order);
      reg1 = __dmul_rd(asin_theta_1d_rtm[l_rtm], reg0);
      reg3 = __dmul_rd(reg4, reg1);         

      sp1 += __dmul_rd(vr_rtm[ip_rtm-3], reg1);
      reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
      reg4 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
      reg3 *= vr_rtm[in_rtm-2];
      reg2 *= vr_rtm[ip_rtm-1];
      sp2 += __dadd_rd(reg0, reg4); 
      sp3 -= __dadd_rd(reg3, reg2); 
    }
    idx_sp += 3; 

    sp_rlm[idx_sp-3] += __dmul_rd(__dmul_rd(r_1d_rlm_r, r_1d_rlm_r), sp1);
    sp_rlm[idx_sp-2] += __dmul_rd(r_1d_rlm_r, sp2);
    sp_rlm[idx_sp-1] += __dmul_rd(r_1d_rlm_r, sp3);

  }
}

__global__
void transF_vec_smem_schmidt(int kst, int *idx_gl_1d_rlm_j, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) {
  extern __shared__ double cache[];
  //dim3 grid(constants.nidx_rlm[1],1,1);
  //dim3 block(constants.nidx_rtm[0],nvec,1);
  int k_rtm = threadIdx.x+kst-1;
  //int j_rlm = blockIdx.x;

// 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;

  double reg0, reg2, reg3, reg4;
  double sp1, sp2, sp3; 
 

  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + blockIdx.x];
//  int degree = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + blockIdx.x];
  double gauss_norm = g_sph_rlm_7[blockIdx.x];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nComp = constants.ncomp;

  int me = threadIdx.x * blockDim.y + threadIdx.y;
  int idx_p_rtm = blockIdx.x*nTheta; 

  while(me < nTheta) {
    reg0 = __dmul_rd(gauss_norm, weight_rtm[me]);
    cache[me] = P_rtm[idx_p_rtm+me] * reg0;
    cache[me+nTheta] = dP_rtm[idx_p_rtm+me] * reg0;
    me += blockDim.x*blockDim.y;
  }
 
  int istep_rtm_t = constants.istep_rtm[1];

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x] - 1;
  ip_rtm = k_rtm * constants.istep_rtm[0];
  int mdx_n = mdx_n_rlm_rtm[blockIdx.x] - 1;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm;

  int idx;
 
  double r_1d_rlm_r = radius_1d_rlm_r[k_rtm]; 

  __syncthreads(); 

    sp1=sp2=sp3=0;
    for(int l_rtm=0; l_rtm<nTheta; l_rtm++) {
      ip_rtm = 3*(threadIdx.y+1) + nComp * (l_rtm * istep_rtm_t + mdx_p); 
      in_rtm = 3*(threadIdx.y+1) + nComp * (l_rtm * istep_rtm_t + mdx_n); 

      reg4 = __dmul_rd(cache[l_rtm], (double) order);
      reg3 = __dmul_rd(asin_theta_1d_rtm[l_rtm], reg4);

      sp1 += __dmul_rd(vr_rtm[ip_rtm-3], cache[l_rtm]);
      reg0 = __dmul_rd(vr_rtm[ip_rtm-2], cache[l_rtm+nTheta]);
      reg4 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
      reg3 *= vr_rtm[in_rtm-2];
      reg2 = __dmul_rd(vr_rtm[ip_rtm-1], cache[l_rtm+nTheta]);
      sp2 += __dadd_rd(reg0, reg4); 
      sp3 -= __dadd_rd(reg3, reg2); 
    }
    int idx_sp = 3*(threadIdx.y+1) + nComp * ( blockIdx.x*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]); 

    sp_rlm[idx_sp-3] += __dmul_rd(__dmul_rd(r_1d_rlm_r, r_1d_rlm_r), sp1);
    sp_rlm[idx_sp-2] += __dmul_rd(r_1d_rlm_r, sp2);
    sp_rlm[idx_sp-1] += __dmul_rd(r_1d_rlm_r, sp3);

}

__global__
void transF_scalar(int kst, double *vr_rtm, double *sp_rlm, double *weight_rtm, int *mdx_p_rlm_rtm, double *P_rtm, double *g_sph_rlm_7, const Geometry_c constants) {
  int k_rtm = threadIdx.x+kst-1;

// 3 for m-1, m, m+1
  unsigned int ip_rtm;

  double sp1=0; 

  double gauss_norm = g_sph_rlm_7[blockIdx.x];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nScalar= constants.nscalar;
  int nComp = constants.ncomp;
  int istep_rtm_r = constants.istep_rtm[0];
  int istep_rtm_t = constants.istep_rtm[1];
  int istep_rtm_m = constants.istep_rtm[2];
  int istep_rlm_r = constants.istep_rlm[0];
  int istep_rlm_j = constants.istep_rlm[1];

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x];
  int idx_p_rtm = blockIdx.x*nTheta; 
  int idx;
 
  for(int t=1; t<=nScalar; t++) {
    sp1 = 0;
    for(int l_rtm=1; l_rtm<=nTheta; l_rtm++) {
      ip_rtm = t + 3*nVector + nComp * ((l_rtm-1) * istep_rtm_t + k_rtm * istep_rtm_r + (mdx_p-1)*istep_rtm_m); 
      idx = idx_p_rtm + l_rtm - 1; 
      sp1 += __dmul_rd(vr_rtm[ip_rtm-1],__dmul_rd(__dmul_rd(gauss_norm, weight_rtm[l_rtm-1]), P_rtm[idx]));
    } 
     
    idx = t + 3*nVector + nComp*((blockIdx.x) * istep_rlm_j + k_rtm*istep_rlm_r); 
    sp_rlm[idx-1] += sp1;
  } 
}

void legendre_f_trans_cuda_(int *ncomp, int *nvector, int *nscalar) {
//  static int nShells = *ked - *kst + 1;
  static int nShells = constants.nidx_rtm[0];
  static int nTheta = constants.nidx_rtm[1];

  dim3 grid(constants.nidx_rlm[1],1,1);
  dim3 block(constants.nidx_rtm[0],1,1);

  constants.ncomp = *ncomp;
  constants.nscalar= *nscalar;
  constants.nvector = *nvector;

//  transF_vec<<<grid, block, 0, streams[0]>>> (1, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);

  dim3 block2(constants.nidx_rtm[0],constants.nvector,1);
  transF_vec_smem_schmidt<<<grid, block2, sizeof(double)*nTheta*2, streams[0]>>> (1, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);
  transF_scalar<<<grid, block, 0, streams[1]>>> (1, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.p_rtm, deviceInput.g_sph_rlm_7, constants);
}
