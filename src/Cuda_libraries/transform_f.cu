#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"

void find_optimal_algorithm_(int *ncomp, int *nvector, int *nscalar) {
  constants.ncomp = *ncomp;
  constants.nscalar= *nscalar;
  constants.nvector = *nvector;

  dim3 grid(constants.nidx_rlm[1],constants.nidx_rtm[0],1);
  dim3 block(constants.nvector, constants.nidx_rtm[0],1);

  Timer wallClock;
  double elapsedTime=0;

  cout << "\tCUDA Fwd vector transform Algorithms: \n"; 
  cout << "nVectors: " << constants.nvector << " nShells: " << constants.nidx_rtm[0] << "\n";

  /*for(int i=0; i<2; i++) {
    wallClock.startTimer();
    switch (i) {
    case naive:
      cout << "\t\t Static implementation with a block size of nShells: ";
	  transF_vec<<<constants.nidx_rlm[1], constants.nidx_rtm[0], 0>>> (1, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);
      break;
    case naive_w_more_threads:
      cout << "\t\t Static implementation with a block size of nVector x nShells: ";
	  transF_vec<<<constants.nidx_rlm[1], block, 0>>> (deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);
      break;
    case reduction:
	  cout << "\t\t Static reduction: ";
	  transF_vec_reduction< 32, 3,
                  hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                      double>
            <<<grid, 32>>> (deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, 
						deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, 
						deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, 
						deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, 
                        deviceInput.g_colat_rtm, deviceInput.p_rtm, 
						deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, 
						deviceInput.asin_theta_1d_rtm, 
                        constants);
  
	  break;

    }
    cudaErrorCheck(hipDeviceSynchronize());
    wallClock.endTimer();
    elapsedTime = wallClock.elapsedTime();
    cout << elapsedTime << "\n"; 
  }*/
}

__global__
void transF_vec(int *idx_gl_1d_rlm_j, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3 grid(constants.nidx_rlm[1],1,1);
  //dim3 block(constants.nvector, constants.nidx_rtm[0],1,1);
  int k_rtm = threadIdx.y;
  //int j_rlm = blockIdx.x;

// 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;

  double reg0, reg1, reg2, reg3, reg4;
  double sp1, sp2, sp3; 

  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + blockIdx.x];
//  int degree = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + blockIdx.x];
  double r_1d_rlm_r = radius_1d_rlm_r[k_rtm]; 

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x] - 1;
  ip_rtm = k_rtm * constants.istep_rtm[0];
  int mdx_n = mdx_n_rlm_rtm[blockIdx.x] - 1;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm;


  int idx;
  int idx_p_rtm = blockIdx.x*constants.nidx_rtm[0]; 
 
  int stride = constants.ncomp * constants.istep_rtm[1];
  int idx_sp = constants.ncomp * ( blockIdx.x*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]); 

  ip_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_p;
  in_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_n;
  sp1=sp2=sp3=0;
  for(int l_rtm=0; l_rtm<constants.nidx_rtm[0]; l_rtm++) {
    idx = idx_p_rtm + l_rtm; 
    
    reg0 = P_rtm[idx] * asin_theta_1d_rtm[l_rtm] * (double) order;
    reg1 = __dmul_rd(vr_rtm[ip_rtm-2], dP_rtm[idx]);
    reg2 = __dmul_rd(vr_rtm[in_rtm-1], reg0);
    reg3 = __dmul_rd(vr_rtm[in_rtm-2], reg0);
    reg4 = __dmul_rd(vr_rtm[ip_rtm-1], dP_rtm[idx]);
    

     sp1 = fma(vr_rtm[ip_rtm-3], P_rtm[idx], sp1);
     sp2 += __dadd_rd(reg1, reg2); 
     sp3 -= __dadd_rd(reg3, reg4); 
     
     ip_rtm +=  stride; 
     in_rtm +=  stride; 
   }
   idx_sp += 3*(threadIdx.x+1); 

   sp_rlm[idx_sp-3] += __dmul_rd(__dmul_rd(r_1d_rlm_r, r_1d_rlm_r), sp1);
   sp_rlm[idx_sp-2] = fma(r_1d_rlm_r, sp2, sp_rlm[idx_sp-2]);
   sp_rlm[idx_sp-1] = fma(r_1d_rlm_r, sp3, sp_rlm[idx_sp-1]);

}

__device__ __forceinline__ void prefetchL1( const double *data, int offset ){

data += offset;

asm("prefetch.global.L1 [%0];"::"l"(data) );

}

/*"The ld.cs load cached streaming operation allocates global lines with evict-first policy in L1 and L2 to limit cache pollution by temporary streaming data that may be accessed once or twice."

Read more at: http://docs.nvidia.com/cuda/parallel-thread-execution/index.html#ixzz3vprmMjjs 
*/

__device__ __forceinline__ double loadCS( const double *data, int offset ){

double variable;

data += offset;

asm("ld.global.f64 %0, [%1];": "=d"(variable) : "l"(data) );

return variable;
}

// Cache at all levels (L1 & L2)
__device__ __forceinline__ double cacheCA( const double *data, int offset ){

double variable;

data += offset;

asm("ld.global.ca.f64 %0, [%1];": "=d"(variable) : "l"(data) );

return variable;
}

__device__
int findSPHId(int *idx_gl_1d_rlm_j, int nModes, int degree) {
  for(int i=0; i<nModes; i++) {
    if(idx_gl_1d_rlm_j[nModes+i] == degree)
      return idx_gl_1d_rlm_j[i];
  }
  return -1;
}
 
__device__
int idSymmetricMode(int *idx_gl_1d_rlm_j, int nModes, int order, int degree) {
  for(int i=0; i<nModes; i++) {
    if(idx_gl_1d_rlm_j[nModes+i] == degree && idx_gl_1d_rlm_j[nModes*2+i] == -1*order)
      return i; 
  }
  return -1;
}

/*__device__ 
void computeLegPoly(int order, int initialDegree, int degree, double theta, double p1, double p2, double *p, double *dp) {
  double reg1, reg2, reg3;
  double dp_;
  for(int itr=initialDegree; itr<degree; itr++) {
    reg1 = __ddiv_rd((double) itr+2-order, (double) itr+2+order); 
    reg2 = __dmul_rd(cos(theta), p2);
    reg1 = __dsqrt_rd(reg1);
    reg2 = __dmul_rd((double) 2*itr + 3, reg2);
    reg3 = (double) (itr+2-order) * (itr+1-order);
    reg1 = __dmul_rd(reg1, reg2);
    reg2 = __dmul_rd((double) order+itr+1, p1);
    //dp_ is a misnomer here
    dp_ = (double) (itr+2+order) * (itr+order+1);
    p1 = p2;
    p2 = __dsqrt_rd(__ddiv_rd(reg3, dp_));
    p2 = __dmul_rd(p2, reg2);
    reg3 = (double) itr-order+2;
    // p1, m, l+2
    p2 = __ddiv_rd(__dadd_rd(reg1,-1*p2), reg3);
    //dp_
    dp_ = __dmul_rd(cos(theta), p1);
    dp_ *= (double) itr+2;
    reg1 = __dmul_rd((double)order - itr - 2,p2);
    reg2 = __dsqrt_rd(__ddiv_rd((double) itr+order+2, (double) itr-order+2));
    dp_ = __dadd_rd(dp_, __dmul_rd(reg2, reg1));
    reg3 = -sin(theta);
    dp_ = __ddiv_rd(dp_, reg3);
  }

  *p = p1; 
  *dp = dp_; 
}
*/

#define STRIDE 16

//__global__ __launch_bounds__(256, 3)
__global__
void transF_vec(int *idx_gl_1d_rlm_j, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3 grid(constants.nidx_rlm[1],1,1);
/*TODO:  //dim3 blockNew(nThreads);
  // Mapping from new block pattern to old block pattern should optimize access pattern?? 
  // In the works!!*/
  //dim3 block(nVector, constants.nidx_rtm[0],1,1);

  extern __shared__ double legendre[];

  //For a given order, the sum is over variable theta.
  // What order am I?
  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + blockIdx.x];
  int degree = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + blockIdx.x];
 
  int symModeIdx;
 
  if(order < 0) return;
  if(order==0) symModeIdx=blockIdx.x;
  else
    symModeIdx = idSymmetricMode(idx_gl_1d_rlm_j,constants.nidx_rlm[1],order,degree); 
  //Unique thread id for a 3D block
  int tId = (threadIdx.z * blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;

  int ip_rtm, in_rtm;

  double reg0, reg1, reg2, reg3;
  //Spectral Data points
  double sp1, sp2, sp3; 

    //Cache modes across thread block.
    //Assuming that the amount of shared memory is equivalent to the number of theta values.
    //TODO: For any given amount of shared memory, the block of threads should simply cache, compute,recache, and so on.
  int chunks = (constants.nidx_rtm[1])/( blockDim.x * blockDim.y * blockDim.z);  
    int rem = (constants.nidx_rtm[1]) % ( blockDim.x * blockDim.y * blockDim.z);

    for(int itr=0; itr < chunks; itr++) {
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId] = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId + constants.nidx_rtm[1]] = dP_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
    }
    //Cache the leftover theta terms.
    if (tId < rem) {
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId] = P_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId + constants.nidx_rtm[1]] = dP_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
    }
    __syncthreads();

/*
  int chunks = (constants.nidx_rtm[1])/( blockDim.x * blockDim.y * blockDim.z);  
  int rem = (constants.nidx_rtm[1]) % ( blockDim.x * blockDim.y * blockDim.z);
 
  int initialDegree=(degree/(STRIDE+2)) * (STRIDE+2);

  //TODO: Implement a smarter search algorithm that takes into account data locality.
  int j1, j2;
  if((initialDegree+1) < degree && degree < (initialDegree+2+STRIDE)) {
    j1 = findSPHId(idx_gl_1d_rlm_j, constants.nidx_rlm[1], initialDegree);
    j2 = findSPHId(idx_gl_1d_rlm_j, constants.nidx_rlm[1], initialDegree+1);
  }

  if ((initialDegree+1) < degree && degree < (initialDegree+2+STRIDE)) {
    for(int itr=0; itr<chunks; itr++) {
      double p1, p2, dp_;
      p1 = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + (j1)*constants.nidx_rtm[1]]; 
      p2 = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + (j2)*constants.nidx_rtm[1]]; 
      double theta = g_colat_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId];;
//      computeLegPoly(order, initialDegree, degree, g_colat_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId], reg1, reg2, &reg3, &reg0);
      for(int deg=initialDegree; deg<degree; deg++) {
        reg1 = __ddiv_rd((double) deg+2-order, (double) deg+2+order); 
        reg2 = __dmul_rd(cos(theta), p2);
        reg1 = __dsqrt_rd(reg1);
        reg2 = __dmul_rd((double) 2*deg + 3, reg2);
        reg3 = (double) (deg+2-order) * (deg+1-order);
        reg1 = __dmul_rd(reg1, reg2);
        reg2 = __dmul_rd((double) order+deg+1, p1);
        //dp_ is a misnomer here
        dp_ = (double) (deg+2+order) * (deg+order+1);
        p1 = p2;
        p2 = __dsqrt_rd(__ddiv_rd(reg3, dp_));
        p2 = __dmul_rd(p2, reg2);
        reg3 = (double) deg-order+2;
        // p1, m, l+2
        p2 = __ddiv_rd(__dadd_rd(reg1,-1*p2), reg3);
      }
      //dp_
      dp_ = __dmul_rd(cos(theta), p1);
      dp_ *= (double) degree;
      reg1 = __dmul_rd((double)order - degree - 3,p2);
      reg2 = __dsqrt_rd(__ddiv_rd((double) degree+order+1, (double) degree-order+1));
      dp_ = __dadd_rd(dp_, __dmul_rd(reg2, reg1));
      reg3 = -sin(theta);
      dp_ = __ddiv_rd(dp_, reg3);
      
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId] = p1; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId + constants.nidx_rtm[1]] = dp_; 
    } 
  }
  else {
    for(int itr=0; itr<chunks; itr++) {
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId] = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId + constants.nidx_rtm[1]] = dP_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
    }
  }

    //Cache the leftover theta terms.
  if (tId < rem) {
    if ((initialDegree+1) < degree && degree < (initialDegree+2+STRIDE)) {
      double p1, p2, dp_;
      double theta = g_colat_rtm[chunks*( blockDim.x * blockDim.y * blockDim.z) + tId];;
      p1 = P_rtm[chunks*( blockDim.x * blockDim.y * blockDim.z) + tId + (j1)*constants.nidx_rtm[1]]; 
      p2 = P_rtm[chunks*( blockDim.x * blockDim.y * blockDim.z) + tId + (j2)*constants.nidx_rtm[1]]; 
      for(int itr=initialDegree+1; itr<degree; itr++) {
        reg1 = __ddiv_rd((double) itr+2-order, (double) itr+2+order); 
        reg2 = __dmul_rd(cos(theta), p2);
        reg1 = __dsqrt_rd(reg1);
        reg2 = __dmul_rd((double) 2*itr + 3, reg2);
        reg3 = (double) (itr+2-order) * (itr+1-order);
        reg1 = __dmul_rd(reg1, reg2);
        reg2 = __dmul_rd((double) order+itr+1, p1);
        //dp_ is a misnomer here
        dp_ = (double) (itr+2+order) * (itr+order+1);
        p1 = p2;
        p2 = __dsqrt_rd(__ddiv_rd(reg3, dp_));
        p2 = __dmul_rd(p2, reg2);
        reg3 = (double) itr-order+2;
        // p1, m, l+2
        p2 = __ddiv_rd(__dadd_rd(reg1,-1*p2), reg3);
      }
      //dp_
      dp_ = __dmul_rd(cos(theta), p1);
      dp_ *= (double) degree;
      reg1 = __dmul_rd((double)order - degree - 3,p2);
      reg2 = __dsqrt_rd(__ddiv_rd((double) degree+order+1, (double) degree-order+1));
      dp_ = __dadd_rd(dp_, __dmul_rd(reg2, reg1));
      reg3 = -sin(theta);
      dp_ = __ddiv_rd(dp_, reg3);
      
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId] = p1; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId + constants.nidx_rtm[1]] = dp_; 
    }
    else {
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId] = P_rtm[chunks*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
      legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId + constants.nidx_rtm[1]] = dP_rtm[chunks*( blockDim.x * blockDim.y * blockDim.z) + tId + blockIdx.x*constants.nidx_rtm[1]]; 
    }
  }
  __syncthreads();
  */
 
//** nVectors*nShells has to be less than or equal to nTheta
  //prefetchL1(weight_rtm, tId);
  //prefetchL1(asin_theta_1d_rtm, tId);

  //Case (nVec * nShells < nThreads) unaccounted for! 

//    vecId = tId % constants.nvector;
//    k_rtm = tId/constants.nvector;
       

//  int k_rtm = threadIdx.y;
//  int idx_p_rtm = blockIdx.x*constants.nidx_rtm[1]; 


  double gauss_norm = g_sph_rlm_7[blockIdx.x];
/*  double gauss_norm;
  if( order==0 ) {
    if(degree == 0)
      gauss_norm=1;
    else
      gauss_norm = (2*degree+1)/(2*degree*(degree+1));
  } else 
    gauss_norm = (2*degree+1)/(4*degree*(degree+1)); 
*/
  int idx_p_rtm = blockIdx.x*constants.nidx_rtm[1];
  int idx_p_rtm_sym = symModeIdx*constants.nidx_rtm[1]; 

  //double p_leg = P_rtm[idx_p_rtm];
  //p_leg = legendre[0];
  //dpdt = legendre[constants.nidx_rtm[1]];
//  double dpdt = dP_rtm[idx_p_rtm];
//  double weight =  weight_rtm[0];
//  double asin_t = asin_theta_1d_rtm[0];

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x] - 1;
  int mdx_n = mdx_n_rlm_rtm[blockIdx.x] - 1;
  ip_rtm = threadIdx.y * constants.istep_rtm[0] ;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm;

  ip_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_p;
  in_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_n;
  int stride = constants.ncomp * constants.istep_rtm[1];
 
  mdx_p = mdx_p_rlm_rtm[symModeIdx] - 1;
  mdx_n = mdx_n_rlm_rtm[symModeIdx] - 1;
  int ip_rtm_sym = threadIdx.y * constants.istep_rtm[0] ;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm_sym;
  mdx_n += ip_rtm_sym;

  ip_rtm_sym = 3*(threadIdx.x+1) + constants.ncomp * mdx_p;
  int in_rtm_sym = 3*(threadIdx.x+1) + constants.ncomp * mdx_n;
 
  double reg4, reg5, reg6, reg7; 
  double sp1_sym, sp2_sym, sp3_sym;

  for(int l_rtm=0; l_rtm<constants.nidx_rtm[1]; l_rtm++) {
    idx_p_rtm++; 
    idx_p_rtm_sym++; 
    reg0 = __dmul_rd(gauss_norm, weight_rtm[l_rtm]);
    reg1 = __dmul_rd(reg0, legendre[l_rtm]);
    reg2 = __dmul_rd(reg0, legendre[l_rtm+constants.nidx_rtm[1]]);
    sp1 = fma(vr_rtm[ip_rtm-3], reg1, sp1);
    sp1_sym = fma(vr_rtm[ip_rtm_sym-3], reg1, sp1_sym);
    reg3 = __dmul_rd(__dmul_rd(asin_theta_1d_rtm[l_rtm], (double) order), reg1);
    
  //  weight = weight_rtm[l_rtm];
  //  asin_t = asin_theta_1d_rtm[l_rtm];
//    p_leg = P_rtm[idx_p_rtm];
  //  p_leg = legendre[l_rtm];
 //   dpdt = dP_rtm[idx_p_rtm];
  //  dpdt = legendre[l_rtm + constants.nidx_rtm[1]];
    
    reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
    reg4 = __dmul_rd(vr_rtm[ip_rtm_sym-2], reg2);

    reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
    reg5 = __dmul_rd(vr_rtm[in_rtm_sym-1], reg3);

    reg2 *= vr_rtm[ip_rtm-1];
    reg3 *= vr_rtm[in_rtm-2];

    reg6 = reg2*vr_rtm[ip_rtm_sym-1];
    reg7 = -1*reg3*vr_rtm[in_rtm_sym-2];

    ip_rtm += stride;
    in_rtm += stride;
    ip_rtm_sym += stride;
    in_rtm_sym += stride;

    sp2 += __dadd_rd(reg0, reg1); 
    sp3 -= __dadd_rd(reg2, reg3); 

    sp2_sym += __dadd_rd(reg4, reg5);
    sp3_sym -+ __dadd_rd(reg6, reg7);
  }

  //reg0 = __dmul_rd(gauss_norm, weight);
  int idx_sp = constants.ncomp * ( blockIdx.x*constants.istep_rlm[1] + threadIdx.y*constants.istep_rlm[0]); 
  int idx_sp_sym = constants.ncomp * ( symModeIdx*constants.istep_rlm[1] + threadIdx.y*constants.istep_rlm[0]); 
  /*reg1 = __dmul_rd(reg0, p_leg);
  reg2 = __dmul_rd(reg0, dpdt);
  sp1 += __dmul_rd(vr_rtm[ip_rtm-3], reg1);
  reg3 = __dmul_rd(__dmul_rd(asin_t, (double) order), reg1);

  idx_sp += 3*(threadIdx.x+1); 
  reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
  reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
  reg2 *=  vr_rtm[ip_rtm-1];
  reg3 *=  vr_rtm[in_rtm-2];
  sp2 += __dadd_rd(reg0, reg1); 
  sp3 -= __dadd_rd(reg2, reg3); 
   */ 
  sp_rlm[idx_sp-3] = fma(__dmul_rd(radius_1d_rlm_r[threadIdx.y],radius_1d_rlm_r[threadIdx.y]), sp1, sp_rlm[idx_sp-3]);
  sp_rlm[idx_sp-2] = fma(radius_1d_rlm_r[threadIdx.y], sp2, sp_rlm[idx_sp-2]);
  sp_rlm[idx_sp-1] = fma(radius_1d_rlm_r[threadIdx.y], sp3, sp_rlm[idx_sp-1]);
  
  sp_rlm[idx_sp_sym-3] = fma(__dmul_rd(radius_1d_rlm_r[threadIdx.y],radius_1d_rlm_r[threadIdx.y]), sp1_sym, sp_rlm[idx_sp_sym-3]);
  sp_rlm[idx_sp_sym-2] = fma(radius_1d_rlm_r[threadIdx.y], sp2_sym, sp_rlm[idx_sp_sym-2]);
  sp_rlm[idx_sp_sym-1] = fma(radius_1d_rlm_r[threadIdx.y], sp3_sym, sp_rlm[idx_sp_sym-1]);
}

__global__ void normalizeLegendre(double *P_rtm, double *dP_rtm, double *g_sph_rlm_7, double *weight_rtm, const Geometry_c constants) {
   // dim3 grid(nidx_rlm[1])
   // dim3 block(nidx_rtm[1],1,1)
    P_rtm[blockIdx.x*constants.nidx_rtm[1] + threadIdx.x] *= g_sph_rlm_7[blockIdx.x] * weight_rtm[threadIdx.x];
    dP_rtm[blockIdx.x*constants.nidx_rtm[1] + threadIdx.x] *= g_sph_rlm_7[blockIdx.x] * weight_rtm[threadIdx.x];
}

__global__ void transF_vec_paired(symmetricModes *pairedList, double *vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) 
{
 //dim3 grid(constants.nPairs,1,1);
  //dim3 block(constants.nvector, constants.nidx_rtm[0],1);

  extern __shared__ double legendre[];
  int k_rtm = threadIdx.y;
  
 // 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;
  
  double reg0, reg1, reg2, reg3;
  double sp1, sp2, sp3;
 
  int tId = (threadIdx.z * blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;

  int order, modeIdx;
  order = pairedList[blockIdx.x].order;
  modeIdx = pairedList[blockIdx.x].positiveModeIdx;

  //Cache modes across thread block.
  //Assuming that the amount of shared memory is equivalent to the number of theta values.
  //TODO: For any given amount of shared memory, the block of threads should simply cache, compute,recache, and so on.
  int chunks = (constants.nidx_rtm[1])/( blockDim.x * blockDim.y * blockDim.z);  
  int rem = (constants.nidx_rtm[1]) % ( blockDim.x * blockDim.y * blockDim.z);

  for(int itr=0; itr < chunks; itr++) {
    legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId] = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + modeIdx*constants.nidx_rtm[1]]; 
    legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId + constants.nidx_rtm[1]] = dP_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + modeIdx*constants.nidx_rtm[1]]; 
  }
  //Cache the leftover theta terms.
  if (tId < rem) {
    legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId] = P_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + modeIdx*constants.nidx_rtm[1]]; 
    legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId + constants.nidx_rtm[1]] = dP_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + modeIdx*constants.nidx_rtm[1]]; 
  }

  __syncthreads();

  
//  #pragma unroll
  for(int i=0; i<2; i++) {
    if(i==1) {
      order = -1*pairedList[blockIdx.x].order;
      modeIdx = pairedList[blockIdx.x].negativeModeIdx;
    }

    double gauss_norm = g_sph_rlm_7[modeIdx]; 
//    double gauss_norm = (2*degree+1)/(4*degree*(degree+1)); 
    double weight = weight_rtm[0];
    double asin_t = asin_theta_1d_rtm[0];
    double p_leg = legendre[0];
    double dpdt = legendre[constants.nidx_rtm[1]];

    int mdx_p = mdx_p_rlm_rtm[modeIdx] - 1;
    int mdx_n = mdx_n_rlm_rtm[modeIdx] - 1;
    ip_rtm = k_rtm * constants.istep_rtm[0] ;
    mdx_p *= constants.istep_rtm[2];
    mdx_n *= constants.istep_rtm[2];
    mdx_p += ip_rtm;
    mdx_n += ip_rtm;

    ip_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_p;
    in_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_n;

    int stride = constants.ncomp * constants.istep_rtm[1];

    sp1=sp2=sp3=0;
    for(int l_rtm=1; l_rtm<constants.nidx_rtm[1]; l_rtm++) {
     sp1 += __dmul_rd(vr_rtm[ip_rtm-3], p_leg);
     reg3 = __dmul_rd(__dmul_rd(asin_t, (double) order), p_leg);
     reg0 = __dmul_rd(vr_rtm[ip_rtm-2], dpdt);
     reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
     dpdt *= vr_rtm[ip_rtm-1];
     reg3 *= vr_rtm[in_rtm-2];
     ip_rtm += stride;
     in_rtm += stride;
     sp2 += __dadd_rd(reg0, reg1);
     sp3 -= __dadd_rd(dpdt, reg3);
     asin_t = asin_theta_1d_rtm[l_rtm];
     p_leg = legendre[l_rtm];
     dpdt = legendre[l_rtm+constants.nidx_rtm[1]];
    }
   
    reg0 = __dmul_rd(gauss_norm, weight);
    double r_1d_rlm_r = radius_1d_rlm_r[k_rtm];
    int idx_sp = constants.ncomp * ( modeIdx*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]);
    sp1 += __dmul_rd(vr_rtm[ip_rtm-3], p_leg);
    reg3 = __dmul_rd(__dmul_rd(asin_t, (double) order), p_leg);

    idx_sp += 3*(threadIdx.x+1);
    reg0 = __dmul_rd(vr_rtm[ip_rtm-2], dpdt);
    reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
    dpdt *= vr_rtm[ip_rtm-1];
    reg3 *= vr_rtm[in_rtm-2];
    double r_1d_sq = __dmul_rd(r_1d_rlm_r, r_1d_rlm_r);
    sp2 += __dadd_rd(reg0, reg1);
    sp3 -= __dadd_rd(dpdt, reg3);

    sp_rlm[idx_sp-3] += __dmul_rd(r_1d_sq, sp1);
    sp_rlm[idx_sp-2] += __dmul_rd(r_1d_rlm_r, sp2);
    sp_rlm[idx_sp-1] += __dmul_rd(r_1d_rlm_r, sp3);
  }    
}

__global__ void transF_vec_unpaired(unsymmetricModes *unpairedList, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) {
 //dim3 grid(constants.nSingletons,1,1);
  //dim3 block(constants.nvector, constants.nidx_rtm[0],1);

  extern __shared__ double legendre[];
  int k_rtm = threadIdx.y;
  
 // 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;
  
  double reg0, reg1, reg2, reg3;
  double sp1, sp2, sp3;
 
  int tId = (threadIdx.z * blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;

  int order = unpairedList[blockIdx.x].order;
  int modeIdx = unpairedList[blockIdx.x].modeIdx;


//  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + blockIdx.x];
//  int degree = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + blockIdx.x];

    //Cache modes across thread block.
    //Assuming that the amount of shared memory is equivalent to the number of theta values.
    //TODO: For any given amount of shared memory, the block of threads should simply cache, compute,recache, and so on.
  int chunks = (constants.nidx_rtm[1])/( blockDim.x * blockDim.y * blockDim.z);  
  int rem = (constants.nidx_rtm[1]) % ( blockDim.x * blockDim.y * blockDim.z);

  for(int itr=0; itr < chunks; itr++) {
    legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId] = P_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + modeIdx*constants.nidx_rtm[1]]; 
    legendre[( blockDim.x * blockDim.y * blockDim.z) * itr + tId + constants.nidx_rtm[1]] = dP_rtm[itr*( blockDim.x * blockDim.y * blockDim.z) + tId + modeIdx*constants.nidx_rtm[1]]; 
  }
  //Cache the leftover theta terms.
  if (tId < rem) {
    legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId] = P_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + modeIdx*constants.nidx_rtm[1]]; 
    legendre[( blockDim.x * blockDim.y * blockDim.z) * chunks + tId + constants.nidx_rtm[1]] = dP_rtm[chunks*( blockDim.z * blockDim.x * blockDim.y ) + tId + modeIdx*constants.nidx_rtm[1]]; 
  }

    __syncthreads();

  double gauss_norm = g_sph_rlm_7[modeIdx];
  double weight = weight_rtm[0];
  double asin_t = asin_theta_1d_rtm[0];
  double p_leg = legendre[0]; 
  double dpdt = legendre[constants.nidx_rtm[1]];

  int mdx_p = mdx_p_rlm_rtm[modeIdx] - 1;
  int mdx_n = mdx_n_rlm_rtm[modeIdx] - 1;
  ip_rtm = k_rtm * constants.istep_rtm[0] ;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm; 
  
  ip_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_p;
  in_rtm = 3*(threadIdx.x+1) + constants.ncomp * mdx_n;
  
  int stride = constants.ncomp * constants.istep_rtm[1];
  
  sp1=sp2=sp3=0;
  
  for(int l_rtm=1; l_rtm<constants.nidx_rtm[1]; l_rtm++) {
    reg0 = __dmul_rd(gauss_norm, weight);
    reg1 = __dmul_rd(reg0, p_leg);
    reg2 = __dmul_rd(reg0, dpdt); 
    sp1 += __dmul_rd(vr_rtm[ip_rtm-3], reg1);
    reg3 = __dmul_rd(__dmul_rd(asin_t, (double) order), reg1);
    
    weight = weight_rtm[l_rtm];
    asin_t = asin_theta_1d_rtm[l_rtm];
    p_leg = legendre[l_rtm];
    dpdt = legendre[l_rtm+constants.nidx_rtm[1]];
    
    reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
    reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
    reg2 *= vr_rtm[ip_rtm-1];
    reg3 *= vr_rtm[in_rtm-2];
    ip_rtm += stride;
    in_rtm += stride;
    sp2 += __dadd_rd(reg0, reg1);
    sp3 -= __dadd_rd(reg2, reg3);
  } 
    
  reg0 = __dmul_rd(gauss_norm, weight);
  double r_1d_rlm_r = radius_1d_rlm_r[k_rtm];
  int idx_sp = constants.ncomp * ( modeIdx*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]);
  reg1 = __dmul_rd(reg0, p_leg);
  reg2 = __dmul_rd(reg0, dpdt);
  sp1 += __dmul_rd(vr_rtm[ip_rtm-3], reg1);
  reg3 = __dmul_rd(__dmul_rd(asin_t, (double) order), reg1);

  idx_sp += 3*(threadIdx.x+1); 
  reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
  reg1 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
  reg2 *= vr_rtm[ip_rtm-1];
  reg3 *= vr_rtm[in_rtm-2];
  double r_1d_sq = __dmul_rd(r_1d_rlm_r, r_1d_rlm_r);
  sp2 += __dadd_rd(reg0, reg1); 
  sp3 -= __dadd_rd(reg2, reg3); 
    

  sp_rlm[idx_sp-3] += __dmul_rd(r_1d_sq, sp1);
  sp_rlm[idx_sp-2] += __dmul_rd(r_1d_rlm_r, sp2);
  sp_rlm[idx_sp-1] += __dmul_rd(r_1d_rlm_r, sp3);
}
 
__global__ void transF_vec_paired_tiny(symmetricModes *pairedList, const int kLoad, double const* vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *P_rtm, double *dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) 
{
 //dim3 grid(constants.nPairs,constants.nidx_rtm[0],1);
  //dim3 block(constants.nvector,1);

  extern __shared__ double legendre[];

  
 // 3 for m-1, m, m+1
  int ip_rtm, in_rtm;
  
  double reg0, reg1, reg2, reg3;
  double sp1, sp2, sp3;
 
  int tId = threadIdx.x;

  int order, modeIdx;
  order = pairedList[blockIdx.x].order;
  modeIdx = pairedList[blockIdx.x].positiveModeIdx;

  //Cache modes across thread block.
  //Assuming that the amount of shared memory is equivalent to the number of theta values.
  //TODO: For any given amount of shared memory, the block of threads should simply cache, compute,recache, and so on.
  int chunks = (constants.nidx_rtm[1])/blockDim.x;  
  int rem = (constants.nidx_rtm[1]) % blockDim.x;

  int stride = modeIdx*constants.nidx_rtm[1] + tId;
  for(int itr=0; itr < chunks; itr++) {
    legendre[blockDim.x * itr + tId] = P_rtm[itr*blockDim.x + stride]; 
    legendre[blockDim.x * itr + tId] = dP_rtm[itr*blockDim.x + stride]; 
  }
  //Cache the leftover theta terms.
  if (tId < rem) {
    legendre[ blockDim.x * chunks + tId] = P_rtm[chunks*blockDim.x + stride]; 
    legendre[ blockDim.x * chunks + tId + constants.nidx_rtm[1]] = dP_rtm[chunks*blockDim.x + stride]; 
  }

  __syncthreads();

  
  for(int i=0; i<2; i++) {
    if(i==1) {
      order = -1*pairedList[blockIdx.x].order;
      modeIdx = pairedList[blockIdx.x].negativeModeIdx;
    }

    // Constant variables over block of threads: 
    double gauss_norm = g_sph_rlm_7[modeIdx]; 
//    double gauss_norm = (2*degree+1)/(4*degree*(degree+1)); 
    int mdx_p = (mdx_p_rlm_rtm[modeIdx] - 1) * constants.istep_rtm[2];
    int mdx_n = (mdx_n_rlm_rtm[modeIdx] - 1) * constants.istep_rtm[2];

    for(int k_rtm=0; k_rtm<kLoad; k_rtm++) {
      k_rtm = blockIdx.y;
      double weight = weight_rtm[0];
      double asin_t = asin_theta_1d_rtm[0];
      double p_leg = legendre[0];
      double dpdt = legendre[constants.nidx_rtm[1]];

      ip_rtm = 3*(threadIdx.x+1) + constants.ncomp * (mdx_p + k_rtm * constants.istep_rtm[0]);
      in_rtm = 3*(threadIdx.x+1) + constants.ncomp * (mdx_n + k_rtm * constants.istep_rtm[0]);

      int stride = constants.ncomp * constants.istep_rtm[1];

      double sp1=sp2=sp3=0;
      for(int l_rtm=0; l_rtm<constants.nidx_rtm[1]; l_rtm++, ip_rtm+=stride, in_rtm+=stride) {
        reg0 = gauss_norm * weight_rtm[l_rtm] * legendre[l_rtm];
        reg1 = gauss_norm * weight_rtm[l_rtm] * legendre[l_rtm+constants.nidx_rtm[1]];
        sp1 += vr_rtm[ip_rtm-3] * reg0;
        sp2 += vr_rtm[ip_rtm-2]*reg1 - vr_rtm[in_rtm-1]*asin_theta_1d_rtm[l_rtm]*order*reg0;
        sp3 -+ vr_rtm[ip_rtm-1]*reg1 + vr_rtm[in_rtm-2]*asin_theta_1d_rtm[l_rtm]*order*reg0;
      }

      double r_1d_rlm_r = radius_1d_rlm_r[k_rtm];
      double r_1d_sq = __dmul_rd(r_1d_rlm_r, r_1d_rlm_r);

      int idx_sp = 3*(threadIdx.x+1) + constants.ncomp * ( modeIdx*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]);

      sp_rlm[idx_sp-3] += r_1d_rlm_r*r_1d_rlm_r*sp1;
      sp_rlm[idx_sp-2] += r_1d_rlm_r*sp2;
      sp_rlm[idx_sp-1] += r_1d_rlm_r*sp3;
    }
  }    
}

//Reduction using an open source library CUB supported by nvidia
/*template <
    int     THREADS_PER_BLOCK,
    int			ITEMS_PER_THREAD,
    hipcub::BlockReduceAlgorithm ALGORITHM,
    typename T>
__global__ void transF_vec_paired(symmetricModes *pairedList, int *idx_gl_1d_rlm_j, double const* __restrict__ vr_rtm, double *sp_rlm, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm, double *a_r_1d_rlm_r, double *g_colat_rtm, double const* __restrict__ P_rtm, double const* __restrict__ dP_rtm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3 grid(constants.nidx_rlm[1],constants.nidx_rtm[0],1); 
  //dim3 block(nThreads,1,1);
  // nThreads * ITEMS_PER_THREAD = nTheta

  typedef hipcub::BlockReduce<T, THREADS_PER_BLOCK, ALGORITHM> BlockReduceT;

  __shared__ typename BlockReduceT::TempStorage temp_storage;

  int k_rtm = blockIdx.y;
  int j_rlm = pairedList[blockIdx.x].positiveModeIdx; 

// 3 for m-1, m, m+1
  unsigned int ip_rtm, in_rtm;

  double reg0, reg1, reg2, reg3, reg4;
  double sp1, sp2, sp3; 

  int order = pairedList[blockIdx.x].order;

  double gauss_norm = g_sph_rlm_7[j_rlm];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nComp = constants.ncomp;

  int mdx_p = mdx_p_rlm_rtm[j_rlm] - 1;
  ip_rtm = k_rtm * constants.istep_rtm[0];
  int mdx_n = mdx_n_rlm_rtm[j_rlm] - 1;
  mdx_p *= constants.istep_rtm[2];
  mdx_n *= constants.istep_rtm[2];
  mdx_p += ip_rtm;
  mdx_n += ip_rtm;

  int idx;
  int idx_p_rtm = j_rlm*nTheta; 
 
  double r_1d_rlm_r = radius_1d_rlm_r[k_rtm]; 
  int idx_sp = nComp * ( blockIdx.x*constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]); 

  double poloidal[ITEMS_PER_THREAD];
  double radial_diff_poloidal[ITEMS_PER_THREAD]; 
  double toroidal[ITEMS_PER_THREAD];

  unsigned int l_rtm=0;

  for(int t=1; t<=nVector; t++) {
    sp1=sp2=sp3=0;
    for(int counter=0; counter < ITEMS_PER_THREAD; counter++) {  
      l_rtm = j_rlm*counter + threadIdx.x; 
      ip_rtm = 3*t + nComp * (l_rtm * constants.istep_rtm[1] + mdx_p); 
      in_rtm = 3*t + nComp * (l_rtm * constants.istep_rtm[1] + mdx_n); 

      idx = idx_p_rtm + l_rtm; 
      reg0 = __dmul_rd(gauss_norm, weight_rtm[l_rtm]);
      reg1 = __dmul_rd(reg0, P_rtm[idx]);
      reg2 = __dmul_rd(reg0, dP_rtm[idx]);
      reg4 = __dmul_rd(P_rtm[idx], (double) order);
      reg1 = __dmul_rd(asin_theta_1d_rtm[l_rtm], reg0);
      reg3 = __dmul_rd(reg4, reg1);         

      poloidal[counter] = __dmul_rd(vr_rtm[ip_rtm-3], reg1);
      reg0 = __dmul_rd(vr_rtm[ip_rtm-2], reg2);
      reg4 =  -1 * __dmul_rd(vr_rtm[in_rtm-1], reg3);
      reg3 *= vr_rtm[in_rtm-2];
      reg2 *= vr_rtm[ip_rtm-1];
      radial_diff_poloidal[counter] = __dadd_rd(reg0, reg4); 
      // After the reduction, toroidal[...] * -1
      toroidal[counter] = __dadd_rd(reg3, reg2); 
    }
    
    idx_sp += 3; 

    __syncthreads();
    sp1 = BlockReduceT(temp_storage).Sum(poloidal);
    __syncthreads();
    sp2 = BlockReduceT(temp_storage).Sum(radial_diff_poloidal);
    __syncthreads();
    sp3 = -1 * BlockReduceT(temp_storage).Sum(toroidal);

    sp_rlm[idx_sp-3] += __dmul_rd(__dmul_rd(r_1d_rlm_r, r_1d_rlm_r), sp1);
    sp_rlm[idx_sp-2] += __dmul_rd(r_1d_rlm_r, sp2);
    sp_rlm[idx_sp-1] += __dmul_rd(r_1d_rlm_r, sp3);
  }
}
*/
__global__
void transF_scalar(int kst, double *vr_rtm, double *sp_rlm, double *weight_rtm, int *mdx_p_rlm_rtm, double *P_rtm, double *g_sph_rlm_7, const Geometry_c constants) {
  int k_rtm = threadIdx.x+kst-1;

// 3 for m-1, m, m+1
  unsigned int ip_rtm;

  double gauss_norm = g_sph_rlm_7[blockIdx.x];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nScalar= constants.nscalar;
  int nComp = constants.ncomp;
  int istep_rtm_r = constants.istep_rtm[0];
  int istep_rtm_t = constants.istep_rtm[1];
  int istep_rtm_m = constants.istep_rtm[2];
  int istep_rlm_r = constants.istep_rlm[0];
  int istep_rlm_j = constants.istep_rlm[1];

  double sp1;
  int mdx_p = mdx_p_rlm_rtm[blockIdx.x];
  int idx_p_rtm = blockIdx.x*nTheta; 
  int idx;
 
  for(int t=1; t<=nScalar; t++) {
    sp1 = 0;
    for(int l_rtm=1; l_rtm<=nTheta; l_rtm++) {
      ip_rtm = t + 3*nVector + nComp * ((l_rtm-1) * istep_rtm_t + k_rtm * istep_rtm_r + (mdx_p-1)*istep_rtm_m); 
      idx = idx_p_rtm + l_rtm - 1; 
      sp1 += __dmul_rd(vr_rtm[ip_rtm-1],__dmul_rd(__dmul_rd(gauss_norm, weight_rtm[l_rtm-1]), P_rtm[idx]));
    } 
     
    idx = t + 3*nVector + nComp*((blockIdx.x) * istep_rlm_j + k_rtm*istep_rlm_r); 
    sp_rlm[idx-1] += sp1;
  } 
}
/*
//Reduction using an open source library CUB supported by nvidia
template <
    int     THREADS_PER_BLOCK,
    int			ITEMS_PER_THREAD,
    hipcub::BlockReduceAlgorithm ALGORITHM,
    typename T>
__global__
void transF_scalar_reduction(double *vr_rtm, double *sp_rlm, double *weight_rtm, int *mdx_p_rlm_rtm, double *P_rtm, double *g_sph_rlm_7, const Geometry_c constants) {
//grid(nidx_rlm[1], nidx_rlm[0])

  typedef hipcub::BlockReduce<T, THREADS_PER_BLOCK, ALGORITHM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp_storage;  

  int k_rtm = blockIdx.y;
  int l_rtm; 

// 3 for m-1, m, m+1
  unsigned int ip_rtm;

  double gauss_norm = g_sph_rlm_7[blockIdx.x];
  int nTheta = constants.nidx_rtm[1];
  int nVector = constants.nvector;
  int nScalar= constants.nscalar;
  int nComp = constants.ncomp;

  int mdx_p = mdx_p_rlm_rtm[blockIdx.x];
  int idx_p_rtm = blockIdx.x*nTheta; 
  int idx;

  double spectral[ITEMS_PER_THREAD]; 

  for(int t=1; t<=nScalar; t++) {
    for(int counter = 0; counter < ITEMS_PER_THREAD; counter ++) {
      l_rtm = blockDim.x*counter + threadIdx.x; 
      ip_rtm = t + 3*nVector + nComp * (l_rtm * constants.istep_rtm[1] + k_rtm * constants.istep_rtm[0] + (mdx_p-1)*constants.istep_rtm[2]); 
      idx = idx_p_rtm + l_rtm; 
	  spectral[counter] = __dmul_rd(vr_rtm[ip_rtm-1],__dmul_rd(__dmul_rd(gauss_norm, weight_rtm[l_rtm]), P_rtm[idx]));
    }
    idx = t + 3*nVector + nComp*((blockIdx.x) * constants.istep_rlm[1] + k_rtm*constants.istep_rlm[0]); 
    __syncthreads();
    sp_rlm[idx-1] = BlockReduceT(temp_storage).Sum(spectral);
  } 
}
*/

void legendre_f_trans_cuda_(int *ncomp, int *nvector, int *nscalar) {
  static int nShells = constants.nidx_rtm[0];

  constants.ncomp = *ncomp;
  constants.nscalar= *nscalar;
  constants.nvector = *nvector;

  dim3 grid(constants.nidx_rlm[1],nShells,1);
  dim3 block(constants.nvector, constants.nidx_rtm[0],1);

  //ToDo: Ponder this: if not exact, what are the consequences?
  //Extremeley important! *****
  //int itemsPerThread = constants.nidx_rtm[1]/blockSize; 
  //std::assert(itemsPerThread*blockSize == constants.nidx_rtm[1]);
  //std::assert(minGridSize <= constants.nidx_rlm[1]);

#ifdef CUDA_TIMINGS
  static Timer transF_v("fwd vector algorithm ");
  cudaPerformance.registerTimer(&transF_v);
  transF_v.startTimer();
#endif
/*  transF_vec_reduction< 32, 3,
                  hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                      double>
            <<<grid, 32>>> (deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, 
                        deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, 
                        deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, 
                        constants);
*/
  transF_vec<<<constants.nidx_rlm[1], block, 0, streams[0]>>> (deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.asin_theta_1d_rtm, constants);
//  transF_vec_unpaired<<<constants.nSingletons, block, 2*sizeof(double)*constants.nidx_rtm[1], streams[0]>>> (deviceInput.unpairedList, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);
//  transF_vec_paired<<<constants.nPairs, block, 2*sizeof(double)*constants.nidx_rtm[1], streams[1]>>> (deviceInput.pairedList, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.radius_1d_rlm_r, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.mdx_n_rlm_rtm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.asin_theta_1d_rtm, constants);
 
#ifdef CUDA_TIMINGS
  cudaDevSync();
  transF_v.endTimer();

  static Timer transF_s("Fwd scalar algorithm ");
  cudaPerformance.registerTimer(&transF_s);
  transF_s.startTimer();
#endif
  /*transF_scalar_reduction <32, 3, 
                     hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                     double>
               <<<grid, 32>>> (deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.p_rtm, deviceInput.g_sph_rlm_7, constants);
*/  
  transF_scalar<<<constants.nidx_rlm[1], nShells, 0, streams[1]>>> (1, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.weight_rtm, deviceInput.mdx_p_rlm_rtm, deviceInput.p_rtm, deviceInput.g_sph_rlm_7, constants);
#ifdef CUDA_TIMINGS
  cudaDevSync();
  transF_s.endTimer();
#endif
  
}
