#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "helper_cublas.h"
//#include "hip/hip_runtime_api.h"

//Using Cuda Registers
__constant__ int indx_rtm;
__constant__ int indx_rlm;


void cublasStatusCheck(hipblasStatus_t stat) {
  if ( strcmp(_cublasGetErrorEnum(stat),"cublasSuccess") != 0 ) {
    printf("%s\n", _cublasGetErrorEnum(stat));
  }
  return;
}
void cudaErrorCheck(hipError_t error) {
  if ( strcmp(_cudaGetErrorEnum(error), "hipSuccess") != 0 ) {
    printf ("%s\n", _cudaGetErrorEnum(error));
  }
  return;
}

__global__ void scaleBy_GaussSphMatrix(hipblasStatus_t *ptrStat, double* Pws_l_d, double* g_sph_rlm_d) {
  __shared__ double scale;

  int me = blockIdx.x;
  int index = indx_rtm*(me);

  scale = g_sph_rlm_d[me];

  hipblasStatus_t stat;
  hipblasHandle_t coHandle;
  stat = hipblasCreate(&coHandle);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    ptrStat = &stat;
    return;
  }
  
  stat = hipblasDscal(coHandle, indx_rtm, &scale, Pws_l_d + index, 1);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    ptrStat = &stat;
    return;
  }

  hipblasDestroy(coHandle);
  ptrStat = &stat;
  return;
}

__global__ void scaleBy_Weights(hipblasStatus_t *ptrStat, double* Pws_l_d, double* weight_rtm_d) {
  __shared__ double scale;

  int me = blockIdx.x;
  scale = weight_rtm_d[me];

  hipblasStatus_t stat;
  hipblasHandle_t coHandle;
  stat = hipblasCreate(&coHandle);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    ptrStat = &stat;
    return;
  }

  stat = hipblasDscal(coHandle, indx_rlm, &scale, Pws_l_d + me, indx_rtm);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    ptrStat = &stat;
    return;
  }

  hipblasDestroy(coHandle);
  ptrStat = &stat;
  return;
}

 
extern "C" void spectral_to_grid_(int* nidx_rtm, int* nidx_rlm, double* P_rtm, double* g_sph_rlm, double* weight_rtm, double* Pws) {
  //Declaring device matrices 
  double *P_rtm_d, *g_sph_rlm_d, *weight_rtm_d;

  //hipProfilerStart();
  //Defining cuda and cublas debugging variables
  hipError_t error;
  hipblasStatus_t stat;

  //Testing cublas
  hipblasHandle_t handle;
  stat = hipblasCreate(&handle);
  cublasStatusCheck(stat);

  // Initializing 2 cuda constant variables:
  // 	1). number of modes 
  //    2). number of meridians

  error = hipMemcpyToSymbol(HIP_SYMBOL(indx_rtm), nidx_rtm, sizeof(int));
  cudaErrorCheck(error);
  error = hipMemcpyToSymbol(HIP_SYMBOL(indx_rlm), nidx_rlm, sizeof(int));
  cudaErrorCheck(error);

  //allocating space for device matrices:
  //P_rtm_d
  error = hipMalloc((void**)&P_rtm_d, sizeof(*P_rtm)*(*nidx_rtm)*(*nidx_rlm));
  cudaErrorCheck(error);
  //Moving data from host to device.
  stat = hipblasSetMatrixAsync(*nidx_rtm, *nidx_rlm, sizeof(*P_rtm), P_rtm, *nidx_rtm, P_rtm_d, *nidx_rtm, 0);
  cublasStatusCheck(stat);

  //allocating space 
  //g_sph_rlm_d
  error = hipMalloc((void**)&g_sph_rlm_d, sizeof(*g_sph_rlm)*(*nidx_rlm));
  cudaErrorCheck(error);
  //Moving data from host to device
  stat = hipblasSetVectorAsync(*nidx_rlm, sizeof(double), g_sph_rlm, 1, g_sph_rlm_d, 1, 0);
  cublasStatusCheck(stat);

  //call to kernel
  scaleBy_GaussSphMatrix<<<*nidx_rlm,1>>> (&stat, P_rtm_d, g_sph_rlm_d);
  //checking if kernel exited safely.
  cublasStatusCheck(stat);
 
  //allocating space
  //weight_rtm_d
  error = hipMalloc((void**)&weight_rtm_d, sizeof(*weight_rtm)*(*nidx_rtm));
  cudaErrorCheck(error);
  //Moving data from host to device
  stat = hipblasSetVectorAsync(*nidx_rtm, sizeof(double), weight_rtm, 1, weight_rtm_d, 1, 0);
  cublasStatusCheck(stat);

  //Synching work from previous call to kernel
  hipDeviceSynchronize();
   
  //call to kernel
  scaleBy_Weights<<<*nidx_rtm, 1>>> (&stat, P_rtm_d, weight_rtm_d);
  cublasStatusCheck(stat);

  //synching threads once more.
  hipDeviceSynchronize();

  //moving device data into a host matrix
  stat = hipblasGetMatrix(*nidx_rtm, *nidx_rlm, sizeof(double), P_rtm_d, *nidx_rtm, Pws, *nidx_rtm);
  cublasStatusCheck(stat);

  hipblasDestroy(handle);

  hipFree(P_rtm_d);
  hipFree(g_sph_rlm_d);  
  hipFree(weight_rtm_d);
//  hipProfilerStop();
//  hipDeviceReset();
  return; 
} 

