#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <sstream>

#include "legendre_poly.h"
//TODO: Use the logger class to write results

void write2file_int_(int *data, int *ncomps, char *fileName, int *nullC) {
  fileName[(*nullC)--] = '\0';
  std::ofstream fp;
  fp.open(fileName);
  for( int i =0; i< (*ncomps); i++) {
    fp <<  data[i] << "\n";
  }
  fp.close();
}
void write2file_(double *data, int *ncomps, char *fileName, int *nullC) {
  fileName[(*nullC)--] = '\0';
  std::ofstream fp;
  fp.open(fileName);
  for( int i =0; i< (*ncomps); i++) {
    fp <<  data[i] << "\n";
  }
  fp.close();
}
 
void check_bwd_trans_cuda_(int *my_rank, double *vr_rtm, double *P_jl, double *dP_jl) {

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
    #if defined(CUDA_OTF)
      std::ofstream schmidt;
    #endif
    std::ofstream field_vec, field_slr;
    //std::string fName = "cuda_schmidt" + "_" + sc.str() + ".log");
    #if defined(CUDA_OTF) 
      fName = "cuda_schmidt_" + sc.str() + ".log";
      schmidt.open(fName.c_str());
    #endif
    fName = "cuda_field_vectors_" + sc.str() + ".log";
    field_vec.open(fName.c_str());
    fName = "cuda_field_scalars_" + sc.str() + ".log";
    field_slr.open(fName.c_str());
      
    #if defined(CUDA_OTF) 
      schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";
    #endif

    double err1=0, err2 = 0, err3 = 0, err4 = 0, err5 = 0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, m, l, mn_rlm;
    int j;
  #if defined(CUDA_OTF) 
    for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++){
      jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
      jed = h_debug.lstack_rlm[mp_rlm];
      //Printing out the ass. legendre polynomials for a single theta value
      int l_rtm=1;
      for(int j_rlm=jst; j_rlm <=jed; j_rlm++) {
        m = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + j_rlm-1];
        l = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1] + j_rlm-1];
        j = l*(l+1) + m;
        pos = (l_rtm-1)*constants.nidx_rlm[1] + l*(l+1) + m;
        schmidt << m << "\t" <<  l << "\t" << P_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1] << "\t" << h_debug.P_smdt[pos]<< "\t" << dP_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1] << "\t" << h_debug.dP_smdt[pos] << "\n"; 
      }
    }
   schmidt.close();
  #endif

   field_vec << "\tshell\tmeridian\tmp_rlm\tvector_index\t vr_rtm_cu[0]\t vr_rtm[0] \t vr_rtm_cu[1] \t vr_rtm[1] \t vr_rtm_cu[2] \t vr_rtm[2] \t vr_rtm_n_cu[0] \t vr_rtm_n[0] \t vr_Rtm_n_cu[1] \t vr_rtm_n[1]\n";
   field_slr << "\tshell\tmeridian\tmp_rlm\tscalar\t vr_rtm_cu[0]\t vr_rtm[0] \n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++) {
        jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
        jed = h_debug.lstack_rlm[mp_rlm];
        mn_rlm = constants.nidx_rtm[2] - mp_rlm + 1;
        for(int l_rtm=1; l_rtm <=constants.nidx_rtm[1]; l_rtm++) {
          for(int nd=1; nd<=constants.nvector; nd++) {
            ip_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            in_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mn_rlm-1)*constants.istep_rtm[2]) - 1;
            err1 = abs(h_debug.vr_rtm[ip_rtm] - vr_rtm[ip_rtm]);
            err2 = abs(h_debug.vr_rtm[ip_rtm-1] - vr_rtm[ip_rtm-1]);
            err3 = abs(h_debug.vr_rtm[ip_rtm-2] - vr_rtm[ip_rtm-2]);
            err4 = abs(h_debug.vr_rtm[in_rtm] - vr_rtm[in_rtm]);
            err5 = abs(h_debug.vr_rtm[in_rtm-1] - vr_rtm[in_rtm-1]);
//            if( err1 >= eps || err2 >= eps || err3 >= eps || err4 >= eps || err5 >= eps)
              field_vec << "\t" << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t"<< h_debug.vr_rtm[ip_rtm-2] << "\t" << vr_rtm[ip_rtm-2] << "\t" << h_debug.vr_rtm[ip_rtm-1] << "\t" << vr_rtm[ip_rtm-1] << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\t" << h_debug.vr_rtm[in_rtm-1] << "\t" << vr_rtm[in_rtm-1] <<"\t" << h_debug.vr_rtm[in_rtm] << "\t" << vr_rtm[in_rtm] << "\n";
          }
          for(int nd=1; nd<=constants.nscalar; nd++) {
            ip_rtm = nd + 3*constants.nvector + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            err1 = abs(h_debug.vr_rtm[ip_rtm] - vr_rtm[ip_rtm]);
  //          if(err1 >= eps)
              field_slr << "\t" << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\n";
          }
        }
      }
    }
   
   field_vec.close();
   field_slr.close();
  #endif
}


void check_fwd_trans_cuda_(int *my_rank, double *sp_rlm) {
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
  #if defined(CUDA_OTF) 
    std::ofstream schmidt;
  #endif
    std::ofstream spec_vec, spec_slr;
  #if defined(CUDA_OTF) 
    fName = "cuda_f_schmidt_" + sc.str() + ".log";
    schmidt.open(fName.c_str());
  #endif
    fName = "cuda_spec_vectors_" + sc.str() + ".log";
    spec_vec.open(fName.c_str());
    fName = "cuda_spec_scalars_" + sc.str() + ".log";
    spec_slr.open(fName.c_str());
      
  #if defined(CUDA_OTF) 
    schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";
    schmidt.close();
  #endif

    double err1=0, err2 = 0, err3 = 0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, order, degree, mn_rlm;
    int i_rlm;
   spec_vec<< "\tshell\tmode\tdegree\torder\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0] \t sp_rlm_cu[1] \t sp_rlm[1] \t sp_rlm_cu[2] \t sp_rlm[2] \n";
   spec_slr<< "\tshell\tmode\tdegree\torder\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0]\n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int j_rlm=1; j_rlm <=constants.nidx_rlm[1]; j_rlm++) {
        degree = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] + (j_rlm-1)];
        order = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] * 2 + (j_rlm-1)];
        for(int nd=1; nd<=constants.nvector; nd++) {
          i_rlm = 3*nd + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
            err1 = abs(h_debug.sp_rlm[i_rlm] - sp_rlm[i_rlm]);
            err2 = abs(h_debug.sp_rlm[i_rlm-1] - sp_rlm[i_rlm-1]);
            err3 = abs(h_debug.sp_rlm[i_rlm-2] - sp_rlm[i_rlm-2]);
   //         if( err1 >= eps || err2 >= eps || err3 >= eps)
              spec_vec << "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t"<< nd << "\t"<< h_debug.sp_rlm[i_rlm-2] << "\t" << sp_rlm[i_rlm-2] << "\t" << h_debug.sp_rlm[i_rlm-1] << "\t" << sp_rlm[i_rlm-1] << "\t" << h_debug.sp_rlm[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
        }
        for(int nd=1; nd<=constants.nscalar; nd++) {
          i_rlm = nd + 3*constants.nvector + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
          err1 = abs(h_debug.sp_rlm[i_rlm] - sp_rlm[i_rlm]);
    //      if( err1 >= eps) 
            spec_slr<< "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t" << nd << "\t" << h_debug.sp_rlm[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
        }
      }
    }

   spec_vec.close();
   spec_slr.close();
  #endif
}

void check_fwd_trans_cuda_and_org_(int *my_rank, double *sp_rlm, double *sp_rlm_debug) {
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
  #if defined(CUDA_OTF) 
    std::ofstream schmidt;
  #endif
    std::ofstream spec_vec, spec_slr;
  #if defined(CUDA_OTF) 
    fName = "cuda_f_schmidt_" + sc.str() + ".log";
    schmidt.open(fName.c_str());
  #endif
    fName = "cuda_spec_vectors_" + sc.str() + ".log";
    spec_vec.open(fName.c_str());
    fName = "cuda_spec_scalars_" + sc.str() + ".log";
    spec_slr.open(fName.c_str());
      
  #if defined(CUDA_OTF) 
    schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";
    schmidt.close();
  #endif

    double err1=0, err2 = 0, err3 = 0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, order, degree, mn_rlm;
    int i_rlm;
   spec_vec<< "\tshell\tmode\tdegree\torder\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0] \t sp_rlm_cu[1] \t sp_rlm[1] \t sp_rlm_cu[2] \t sp_rlm[2] \n";
   spec_slr<< "\tshell\tmode\tdegree\torder\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0]\n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int j_rlm=1; j_rlm <=constants.nidx_rlm[1]; j_rlm++) {
        degree = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] + (j_rlm-1)];
        order = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] * 2 + (j_rlm-1)];
        for(int nd=1; nd<=constants.nvector; nd++) {
          i_rlm = 3*nd + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
            err1 = abs(sp_rlm_debug[i_rlm] - sp_rlm[i_rlm]);
            err2 = abs(sp_rlm_debug[i_rlm-1] - sp_rlm[i_rlm-1]);
            err3 = abs(sp_rlm_debug[i_rlm-2] - sp_rlm[i_rlm-2]);
   //         if( err1 >= eps || err2 >= eps || err3 >= eps)
              spec_vec << "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t"<< nd << "\t"<< sp_rlm_debug[i_rlm-2] << "\t" << sp_rlm[i_rlm-2] << "\t" << sp_rlm_debug[i_rlm-1] << "\t" << sp_rlm[i_rlm-1] << "\t" << sp_rlm_debug[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
        }
        for(int nd=1; nd<=constants.nscalar; nd++) {
          i_rlm = nd + 3*constants.nvector + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
          err1 = abs(h_debug.sp_rlm[i_rlm] - sp_rlm[i_rlm]);
    //      if( err1 >= eps) 
            spec_slr<< "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t" << nd << "\t" << h_debug.sp_rlm[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
        }
      }
    }

   spec_vec.close();
   spec_slr.close();
  #endif
}

void output_spectral_data_cuda_(int *my_rank, int *ncomp, int *nvector, int *nscalar) {
    static bool init = true;
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
    std::ofstream spec_vec, spec_slr;
    if(init) {
      fName = "convergenceBeforeSHT_4Vector_PID_" + sc.str() + ".log";
    }
    else
      fName = "convergenceAfterSHT_4Vector_PID_" + sc.str() + ".log";
    spec_vec.open(fName.c_str());

    if(init) {
      fName = "convergenceBeforeSHT_4Scalar_PID_" + sc.str() + ".log";
      init = false;
    }
    else
      fName = "convergenceAfterSHT_4Scalar_PID_" + sc.str() + ".log";

    spec_slr.open(fName.c_str());
      
    double error=0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, m, l, mn_rlm;
    int i_rlm;
    spec_vec<< "\t shell\tmode\tdegree\torder\tvectorID\t sp_rlm_cu[0]\t sp_rlm_cu[1] \t sp_rlm_cu[2] \n";
    spec_slr<< "\t shell\tmode\tdegree\torder\tvectorID\t sp_rlm_cu[0]\n";

    int order, degree;

    double err1=0, err2=0, err3=0;
    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int j_rlm=1; j_rlm <=constants.nidx_rlm[1]; j_rlm++) {
        degree = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] + (j_rlm-1)];
        order = h_debug.idx_gl_1d_rlm_j[ constants.nidx_rlm[1] * 2 + (j_rlm-1)];
        for(int nd=1; nd<=(*nvector); nd++) {
          i_rlm = 3*nd + (*ncomp)*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
          spec_vec << "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t" << nd << "\t"<< h_debug.sp_rlm[i_rlm-2] << "\t" << h_debug.sp_rlm[i_rlm-1] << "\t" << h_debug.sp_rlm[i_rlm] << "\n";
        }
        for(int nd=1; nd<= (*nscalar); nd++) {
          i_rlm = nd + 3*(*nvector) + (*ncomp)*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
          spec_slr<< "\t" << k << "\t" << j_rlm << "\t" << degree << "\t" << order << "\t" << nd << "\t" << h_debug.sp_rlm[i_rlm] << "\n";
        }
      }
    }
   spec_vec.close();
   spec_slr.close();
}
