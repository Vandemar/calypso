#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include <math.h>
#include <sstream>

void check_bwd_trans_cuda_(int *my_rank, double *vr_rtm, double *P_jl, double *dP_jl) {

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
    #if defined(CUDA_OTF)
      std::ofstream schmidt;
    #endif
    std::ofstream field_vec, field_slr;
    //std::string fName = "cuda_schmidt" + "_" + sc.str() + ".log");
    #if defined(CUDA_OTF) 
      fName = "cuda_schmidt_" + sc.str() + ".log";
      schmidt.open(fName.c_str());
    #endif
    fName = "cuda_field_vectors_" + sc.str() + ".log";
    field_vec.open(fName.c_str());
    fName = "cuda_field_scalars_" + sc.str() + ".log";
    field_slr.open(fName.c_str());
      
    #if defined(CUDA_OTF) 
      schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";
    #endif

    double error=0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, m, l, mn_rlm;
    int j;
  #if defined(CUDA_OTF) 
    for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++){
      jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
      jed = h_debug.lstack_rlm[mp_rlm];
      //Printing out the ass. legendre polynomials for a single theta value
      int l_rtm=1;
      for(int j_rlm=jst; j_rlm <=jed; j_rlm++) {
        m = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + j_rlm-1];
        l = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1] + j_rlm-1];
        j = l*(l+1) + m;
        pos = (l_rtm-1)*constants.nidx_rlm[1] + l*(l+1) + m;
        schmidt << m << "\t" <<  l << "\t" << P_jl[constants.nidx_rlm[1]*(l_rtm-1) + j] << "\t" << h_debug.P_smdt[pos]<< "\t" << dP_jl[constants.nidx_rlm[1]*(l_rtm-1) + j] << "\t" << h_debug.dP_smdt[pos] << "\n"; 
      }
    }
   schmidt.close();
  #endif

   field_vec << "shell\tmeridian\tmp_rlm\tvector_index\t vr_rtm_cu[0]\t vr_rtm[0] \t vr_rtm_cu[1] \t vr_rtm[1] \t vr_rtm_cu[2] \t vr_rtm[2] \t vr_rtm_n_cu[0] \t vr_rtm_n[0] \t vr_Rtm_n_cu[1] \t vr_rtm_n[1]\n";
   field_slr << "shell\tmeridian\tmp_rlm\tscalar\t vr_rtm_cu[0]\t vr_rtm[0] \n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++) {
        jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
        jed = h_debug.lstack_rlm[mp_rlm];
        mn_rlm = constants.nidx_rtm[2] - mp_rlm + 1;
        for(int l_rtm=1; l_rtm <=constants.nidx_rtm[1]; l_rtm++) {
          for(int nd=1; nd<=constants.nvector; nd++) {
            ip_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            in_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mn_rlm-1)*constants.istep_rtm[2]) - 1;
            if(h_debug.vr_rtm[ip_rtm] != vr_rtm[ip_rtm] || h_debug.vr_rtm[ip_rtm-1] != vr_rtm[ip_rtm-1] || h_debug.vr_rtm[ip_rtm-2] != vr_rtm[ip_rtm-2] || h_debug.vr_rtm[in_rtm] != vr_rtm[in_rtm] || h_debug.vr_rtm[in_rtm-1] != vr_rtm[in_rtm-1])
              field_vec << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t"<< h_debug.vr_rtm[ip_rtm-2] << "\t" << vr_rtm[ip_rtm-2] << "\t" << h_debug.vr_rtm[ip_rtm-1] << "\t" << vr_rtm[ip_rtm-1] << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\t" << h_debug.vr_rtm[in_rtm-1] << "\t" << vr_rtm[in_rtm-1] <<"\t" << h_debug.vr_rtm[in_rtm] << "\t" << vr_rtm[in_rtm] << "\n";
          }
          for(int nd=1; nd<=constants.nscalar; nd++) {
            ip_rtm = nd + 3*constants.nvector + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            if(h_debug.vr_rtm[ip_rtm] != vr_rtm[ip_rtm])
              field_slr << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\n";
          }
        }
      }
    }
   
   field_vec.close();
   field_slr.close();
  #endif
}


void check_fwd_trans_cuda_(int *my_rank, double *sp_rlm) {
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::string fName;
    std::stringstream sc;
    sc << *my_rank;
  #if defined(CUDA_OTF) 
    std::ofstream schmidt;
  #endif
    std::ofstream spec_vec, spec_slr;
  #if defined(CUDA_OTF) 
    fName = "cuda_f_schmidt_" + sc.str() + ".log";
    schmidt.open(fName.c_str());
  #endif
    fName = "cuda_spec_vectors_" + sc.str() + ".log";
    spec_vec.open(fName.c_str());
    fName = "cuda_spec_scalars_" + sc.str() + ".log";
    spec_slr.open(fName.c_str());
      
  #if defined(CUDA_OTF) 
    schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";
    schmidt.close();
  #endif

    double error=0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, m, l, mn_rlm;
    int i_rlm;
   spec_vec<< "shell\tmode\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0] \t sp_rlm_cu[1] \t sp_rlm[1] \t sp_rlm_cu[2] \t sp_rlm[2] \n";
   spec_slr<< "shell\tmode\tvector_index\t sp_rlm_cu[0]\t sp_rlm[0]\n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int j_rlm=1; j_rlm <=constants.nidx_rlm[1]; j_rlm++) {
        for(int nd=1; nd<=constants.nvector; nd++) {
          i_rlm = 3*nd + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
            if(h_debug.sp_rlm[i_rlm] != sp_rlm[i_rlm] || h_debug.sp_rlm[i_rlm-1] != sp_rlm[i_rlm-1] || h_debug.sp_rlm[i_rlm-2] != sp_rlm[i_rlm-2] ) {
              spec_vec << k << "\t" << j_rlm << "\t" << nd << "\t"<< h_debug.sp_rlm[i_rlm-2] << "\t" << sp_rlm[i_rlm-2] << "\t" << h_debug.sp_rlm[i_rlm-1] << "\t" << sp_rlm[i_rlm-1] << "\t" << h_debug.sp_rlm[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
          }
        }
        for(int nd=1; nd<=constants.nscalar; nd++) {
          i_rlm = nd + 3*constants.nvector + constants.ncomp*((j_rlm-1)*constants.istep_rlm[1] + (k-1)*constants.istep_rlm[0]) - 1;
          if(h_debug.sp_rlm[i_rlm] != sp_rlm[i_rlm]) {
            spec_slr<< k << "\t" << j_rlm << "\t" << nd << "\t" << h_debug.sp_rlm[i_rlm] << "\t" << sp_rlm[i_rlm] << "\n";
          }
        }
      }
    }
   
   spec_vec.close();
   spec_slr.close();
  #endif
}
