#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <sstream>


void initDevConstVariables() {
//  hipError_t error;
//  error = hipMemcpyToSymbol(HIP_SYMBOL(lstack_rlm), &constants, sizeof(Geometry_c), 0, hipMemcpyHostToDevice);
//  cudaErrorCheck(error);
}

template <
    int     THREADS_PER_BLOCK,
    hipcub::BlockReduceAlgorithm ALGORITHM,
    typename T>
__global__
void transB_scalar_reduction(int *lstack_rlm, double *vr_rtm, double const* __restrict__ sp_rlm, double *P_jl, const Geometry_c constants) {
  //dim3  grid(nidx_rtm[1], nidx_rtm[0])
  //dim3 block(t_lvl+1)

  typedef hipcub::BlockReduce<T, THREADS_PER_BLOCK, ALGORITHM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp_storage;

  int l_rtm = blockIdx.x;
  int jst, jed, j_rlm;

  unsigned int idx_p_jl=0, idx=0, idx_rtm=0;

  double vrs1;
  double P_smdt;
 
  int reg1 = 3*constants.nvector + constants.ncomp*blockIdx.y*constants.istep_rlm[0];
  int reg2 = 3*constants.nvector + constants.ncomp * ((blockIdx.x) * constants.istep_rtm[1] + blockIdx.y*constants.istep_rtm[0]); 

  for(int mp_rlm=0; mp_rlm < constants.nidx_rtm[2]; mp_rlm++) { 
	jst = lstack_rlm[mp_rlm] + 1;
	jed = lstack_rlm[mp_rlm+1];
    int totalWorkLoad = jed-jst+1;
    int threadWorkLoad = totalWorkLoad/THREADS_PER_BLOCK; 
    if( totalWorkLoad % THREADS_PER_BLOCK < threadIdx.x )
      threadWorkLoad++;
    // threadWorkLoad is a negative number... jed is 1 and jst > 1
    // Block (0,5,0) && mp_rlm = 5
    int workingThreads = min(totalWorkLoad, blockDim.x);

    if (threadIdx.x < workingThreads) {
	  j_rlm = jst-1 + threadIdx.x;
      idx_p_jl = constants.nidx_rlm[1]*l_rtm;
      for(int t=1; t<=constants.nscalar; t++) {
        j_rlm = jst-1 + threadIdx.x;
        vrs1=0;
        for( int counter = 0; counter < threadWorkLoad; j_rlm += blockDim.x, counter++) {
          idx = reg1 + t + constants.ncomp*j_rlm*constants.istep_rlm[1]; 
          P_smdt = P_jl[idx_p_jl + j_rlm]; 
          vrs1 += sp_rlm[idx - 1] * P_smdt;
        }
        idx_rtm = reg2 + t + mp_rlm * constants.istep_rtm[2]; 
        __syncthreads();
        vr_rtm[idx_rtm - 1] = BlockReduceT(temp_storage).Sum(vrs1, workingThreads); 
      }
    }
  }
}

__global__
void transB_scalar(int *lstack_rlm, double *vr_rtm, double const* __restrict__ sp_rlm, double *P_jl, Geometry_c constants) {
  //dim3 grid3(nTheta, constants.nidx_rtm[2]);
  //dim3 block3(nShells,1,1);
 // mp_rlm is the blockIdx.y 
  double vrs1;

  int jst = lstack_rlm[blockIdx.y] + 1;
  int jed = lstack_rlm[blockIdx.y+1];
  int idx_p_jl=0, idx=0, idx_rtm=0; 
  int reg1 = 3*constants.nvector + constants.ncomp*threadIdx.x*constants.istep_rlm[0];

  for(int t=1; t<=constants.nscalar; t++) {
    vrs1 = 0;
    idx_p_jl = constants.nidx_rlm[1]*blockIdx.x+jst-1;
    for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
      idx = reg1 + t + constants.ncomp*(j_rlm-1)*constants.istep_rlm[1]; 
      vrs1 += sp_rlm[idx - 1] * P_jl[idx_p_jl];
      idx_p_jl++;
    } 
      
    idx_rtm = t + 3*constants.nvector + constants.ncomp*((blockIdx.x) * constants.istep_rtm[1] + threadIdx.x*constants.istep_rtm[0] + (blockIdx.y)*constants.istep_rtm[2]); 
    vr_rtm[idx_rtm - 1] = vrs1;
  } 
}

//Reduction using an open source library CUB supported by nvidia
template <
    int     THREADS_PER_BLOCK,
    hipcub::BlockReduceAlgorithm ALGORITHM,
    typename T>
__global__
void transB_dydt_reduction(int *lstack_rlm, int *idx_gl_1d_rlm_j, double *vr_rtm, double const* __restrict__ sp_rlm, double *g_sph_rlm, double *a_r_1d_rlm_r, double *P_jl, double *dP_jl, const Geometry_c constants) {
  //dim3  grid(nTheta, nidx_rtm[0])
  //dim3 block(nThreads)

  typedef hipcub::BlockReduce<T, THREADS_PER_BLOCK, ALGORITHM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp_storage;

  int l_rtm = blockIdx.x;
  int k_rtm = blockIdx.y;
  unsigned int idx_p_jl=0, idx=0, idx_rtm=0;

  int j_rlm = 0;

  double a_r_1d_rlm_r_ = a_r_1d_rlm_r[blockIdx.y];
  double vr1, vr2, vr3;

  for(int mp_rlm = 0; mp_rlm < constants.nidx_rtm[2]; mp_rlm++) { 
    int jst = lstack_rlm[mp_rlm] + 1;
	int jed = lstack_rlm[mp_rlm+1];
    int totalWorkLoad = jed-jst+1;
    int threadWorkLoad = totalWorkLoad/THREADS_PER_BLOCK; 
    if( totalWorkLoad % THREADS_PER_BLOCK < threadIdx.x )
      threadWorkLoad++;
    int workingThreads = min(totalWorkLoad, blockDim.x);

    if (threadIdx.x < workingThreads) {
      for(int t=1; t<=constants.nvector; t++) {
        j_rlm = jst - 1 + threadIdx.x;
        vr1=vr2=vr3=0;
		for(int counter = 0; counter < totalWorkLoad; j_rlm += blockDim.x, counter++) {
		  idx = 3*t + constants.ncomp * (j_rlm * constants.istep_rlm[1] + k_rtm * constants.istep_rlm[0]); 
		  idx_p_jl = constants.nidx_rlm[1]*l_rtm+j_rlm;
		  vr3 += sp_rlm[idx - 3] * __dmul_rd(a_r_1d_rlm_r_, a_r_1d_rlm_r_) * P_jl[idx_p_jl] * g_sph_rlm[j_rlm];    
		  vr2 += sp_rlm[idx - 2]  * a_r_1d_rlm_r_ * dP_jl[idx_p_jl];    
		  vr1 -= sp_rlm[idx - 1] * a_r_1d_rlm_r_ * dP_jl[idx_p_jl];    
        }

        idx_rtm = 3*t + constants.ncomp * (l_rtm * constants.istep_rtm[1] + k_rtm*constants.istep_rtm[0] + mp_rlm * constants.istep_rtm[2]); 
      
	   __syncthreads();
	   vr_rtm[idx_rtm - 2 - 1]  += BlockReduceT(temp_storage).Sum(vr3, workingThreads); 
	   __syncthreads();
	    vr_rtm[idx_rtm - 1 - 1]  += BlockReduceT(temp_storage).Sum(vr2, workingThreads); 
	   __syncthreads();
	   vr_rtm[idx_rtm - 1]  += BlockReduceT(temp_storage).Sum(vr1, workingThreads); 
     }
    }
  }
}

__global__
void transB_dydt_old(double *g_sph_rlm, double *vr_rtm, double const* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *P_jl, double *dP_jl, const Geometry_c constants) {
  //dim3 grid3(nTheta, constants.nidx_rtm[2]);
  //dim3 block3(nShells,1,1);

  int mp_rlm = blockIdx.y;
  double a_r_1d = a_r_1d_rlm_r[threadIdx.x];
  int jst = lstack_rlm_cmem[mp_rlm]+1;

  double vr1, vr2, vr3;
  int idx_p_jl=0, idx=0, idx_rtm=0; 
  
  int jed = lstack_rlm_cmem[mp_rlm+1];
  double a_r_1d_rlm_r_sq =  __dmul_rd(a_r_1d, a_r_1d);

  int reg1, reg2;
  double dreg1, dreg2;
  for(int t=1; t<=constants.nvector; t++) {
    reg1 = constants.nidx_rlm[1]*blockIdx.x;
    reg2 = jst-1;
    vr1=vr2=vr3=0;
    idx_p_jl = reg1 + reg2;
    for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
      idx = 3*t;
      reg1 = constants.ncomp * (j_rlm-1) * constants.istep_rlm[1];
      reg2 = constants.ncomp * threadIdx.x * constants.istep_rlm[0]; 
      idx += reg1 + reg2;
      dreg1 = __dmul_rn(P_jl[idx_p_jl], g_sph_rlm[j_rlm]);
      dreg2 = __dmul_rn(a_r_1d, dP_jl[idx_p_jl]);
      vr2 += __dmul_rn(sp_rlm[idx - 2], dreg2);    
      dreg1 *= a_r_1d_rlm_r_sq; 
      vr1 -= __dmul_rn(sp_rlm[idx - 1] , dreg2);
      idx_p_jl++;
      vr3 += __dmul_rn(sp_rlm[idx - 3] , dreg1);
    }
    idx_rtm = 3*t + constants.ncomp * ((blockIdx.x) * constants.istep_rtm[1] + threadIdx.x*constants.istep_rtm[0] + (mp_rlm) * constants.istep_rtm[2]); 

    vr_rtm[idx_rtm - 2 - 1]  += vr3; 
    vr_rtm[idx_rtm - 1 - 1]  += vr2; 
    vr_rtm[idx_rtm - 1]  += vr1; 
  }
}

__global__
void transB_dydt(int *lstack_rlm, int *idx_gl_1d_rlm_j, double *vr_rtm, double const* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *P_jl, double *dP_jl, const Geometry_c constants) {
  //dim3 grid3(nTheta, constants.nidx_rtm[2]);
  //dim3 block3(nShells,1,1);

  int mp_rlm = blockIdx.y;
  int jst = lstack_rlm[mp_rlm] + 1;
  int jed = lstack_rlm[mp_rlm+1];

  double vr1, vr2, vr3;
  unsigned int idx_p_jl=0, idx=0, idx_rtm=0; 
  int deg = idx_gl_1d_rlm_j[constants.nidx_rlm[1] + jst -1];
  int ord = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + jst -1];

  //turn this into a terinary operator
  float g_sph_rlm=deg*(deg+1);
  if (ord==0 && deg==0)
    g_sph_rlm=0.5;

  for(int t=1; t<=constants.nvector; t++) {
    vr1=vr2=vr3=0;
    idx_p_jl = constants.nidx_rlm[1]*blockIdx.x+jst-1;
    for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
      idx = 3*t + constants.ncomp * ((j_rlm-1) * constants.istep_rlm[1] + threadIdx.x * constants.istep_rlm[0]); 
      vr3 += sp_rlm[idx - 3] * __dmul_rd(a_r_1d_rlm_r[threadIdx.x], a_r_1d_rlm_r[threadIdx.x]) * P_jl[idx_p_jl] * g_sph_rlm;    
      vr2 += sp_rlm[idx - 2]  * a_r_1d_rlm_r[threadIdx.x] * dP_jl[idx_p_jl];    
      vr1 -= sp_rlm[idx - 1] * a_r_1d_rlm_r[threadIdx.x] * dP_jl[idx_p_jl];    
      idx_p_jl++;
    }
    idx_rtm = 3*t + constants.ncomp * ((blockIdx.x) * constants.istep_rtm[1] + threadIdx.x*constants.istep_rtm[0] + (mp_rlm) * constants.istep_rtm[2]); 

    vr_rtm[idx_rtm - 2 - 1]  += vr3; 
    vr_rtm[idx_rtm - 1 - 1]  += vr2; 
    vr_rtm[idx_rtm - 1]  += vr1; 
  }
}

template <
    int     THREADS_PER_BLOCK,
    hipcub::BlockReduceAlgorithm ALGORITHM,
    typename T>
__global__
void transB_dydp_reduction(int *lstack_rlm, int *idx_gl_1d_rlm_j, double *vr_rtm, double const* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *P_jl, double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3  grid(nTheta, nidx_rtm[0])
  //dim3 block(nThreads)

  typedef hipcub::BlockReduce<T, THREADS_PER_BLOCK, ALGORITHM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp_storage;

  int j_rlm = 0;
  int k_rtm = blockIdx.y;
  int l_rtm = blockIdx.x;

  unsigned int idx_p_jl=0, idx=0, idx_rtm=0;
  int ord;

  double a_r_1d_rlm_r_ = a_r_1d_rlm_r[k_rtm];

  double vr4, vr5;
  double reg2;

  double reg1 = __dmul_rd(a_r_1d_rlm_r_, asin_theta_1d_rtm[l_rtm]);

  for(int mp_rlm = 0; mp_rlm < constants.nidx_rtm[2]; mp_rlm++) {
    int mn_rlm = constants.nidx_rtm[2] - mp_rlm;
	int jst = lstack_rlm[mp_rlm] + 1;
	int jed = lstack_rlm[mp_rlm+1];
    int totalWorkLoad = jed-jst+1;
    int threadWorkLoad = totalWorkLoad/THREADS_PER_BLOCK; 
    if( totalWorkLoad % THREADS_PER_BLOCK < threadIdx.x )
      threadWorkLoad++;
    int workingThreads = min(totalWorkLoad, blockDim.x);

    ord = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + jst -1];

    if (threadIdx.x < workingThreads) {
      for(int t=1; t<=constants.nvector; t++) {
        vr4=vr5=0;
        j_rlm = jst - 1 + threadIdx.x; 
        for(int counter = 0; counter < workingThreads; counter++, j_rlm += blockDim.x) {
          idx = 3*t + constants.ncomp * (j_rlm * constants.istep_rlm[1] + blockIdx.y * constants.istep_rlm[0]); 
		  idx_p_jl = constants.nidx_rlm[1]*blockIdx.x+j_rlm;
		  reg2 = -1 * __dmul_rd( P_jl[idx_p_jl], __dmul_rd(reg1,(double) ord));         
          vr5 += sp_rlm[idx - 1] * reg2;
          vr4 += sp_rlm[idx - 2] * reg2;        
        }

        idx_rtm = 3*t + constants.ncomp * (l_rtm * constants.istep_rtm[1] + k_rtm*constants.istep_rtm[0] + (mn_rlm - 1) * constants.istep_rtm[2]); 
      
        __syncthreads();
        vr_rtm[idx_rtm - 1 - 1]  += BlockReduceT(temp_storage).Sum(vr5, workingThreads ); 
        __syncthreads();
        vr_rtm[idx_rtm - 1]  += BlockReduceT(temp_storage).Sum(vr4, workingThreads ); 
      }
    }
  }
}

//When looking at the transformed field data, the first component is off by a sign, oddly. 
__global__
void transB_dydp(int *lstack_rlm, int *idx_gl_1d_rlm_j, double *vr_rtm, double const* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *P_jl,  double *asin_theta_1d_rtm, const Geometry_c constants) {
  //dim3 grid3(nTheta, constants.nidx_rtm[2]);
  //dim3 block3(nShells,1,1);
  extern __shared__ double cache[];
  unsigned int idx=0, idx_rtm=0;
  double reg2;
  double vr4, vr5;

  int mn_rlm = constants.nidx_rtm[2] - blockIdx.y;
  int jst = lstack_rlm[blockIdx.y] + 1;
  int jed = lstack_rlm[blockIdx.y+1];
  int order = idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + jst -1]; 
  double asin = asin_theta_1d_rtm[blockIdx.x];
  int idx_p_jl=0; 

  cache[threadIdx.x] = a_r_1d_rlm_r[threadIdx.x] * order * asin;
  __syncthreads();

  for(int t=1; t<=constants.nvector; t++) {
    vr4=vr5=0;
    idx_p_jl = constants.nidx_rlm[1]*blockIdx.x+jst-1;
    for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
      idx = 3*t + constants.ncomp * ((j_rlm-1) * constants.istep_rlm[1] + threadIdx.x * constants.istep_rlm[0]); 
      reg2 = -1 * __dmul_rd( P_jl[idx_p_jl], cache[threadIdx.x]);         
      vr5 += sp_rlm[idx - 1] * reg2;
      vr4 += sp_rlm[idx - 2] * reg2;
      idx_p_jl++;
    }
    // mn_rlm
    idx_rtm = 3*t + constants.ncomp * ((blockIdx.x) * constants.istep_rtm[1] + threadIdx.x*constants.istep_rtm[0] + (mn_rlm-1) * constants.istep_rtm[2]); 

    vr_rtm[idx_rtm - 1 - 1] += vr5; 
    vr_rtm[idx_rtm - 1] += vr4; 
  }
}

void legendre_b_trans_cuda_(int *ncomp, int *nvector, int *nscalar) {
  
//  static int nShells = *ked - *kst + 1;
  static int nShells = constants.nidx_rtm[0];
  static int nTheta = constants.nidx_rtm[1];
 
  constants.ncomp = *ncomp;
  constants.nvector = *nvector;
  constants.nscalar = *nscalar;

  dim3 grid(nTheta, constants.nidx_rtm[2]);

  //The number of threads is an arbitrary value that will vary the amount of thread divergence, the amount of work per thread, and in turn the time efficiency. 

  static Timer transBwdVec_dy_dt("Bwd Vector dydt Transform");
  cudaPerformance.registerTimer(&transBwdVec_dy_dt);
  transBwdVec_dy_dt.startTimer();
  /*transB_dydt_reduction<32, 
                      hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                      double>
                <<<grid, 32>>> (deviceInput.lstack_rlm, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.g_sph_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.p_jl, deviceInput.dP_jl, constants);*/
  
  transB_dydt_old<<<grid, nShells>>> (deviceInput.g_sph_rlm, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.p_jl, deviceInput.dP_jl, constants);
  cudaDevSync();
  transBwdVec_dy_dt.endTimer();

  static Timer transBwdVec_dy_dp("Bwd Vector dydp Transform");
  cudaPerformance.registerTimer(&transBwdVec_dy_dp);
  transBwdVec_dy_dp.startTimer();
  /*transB_dydp_reduction<32, 
                      hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                      double>
                <<<grid, 32>>> (deviceInput.lstack_rlm, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.p_jl, deviceInput.asin_theta_1d_rtm, constants);*/

  transB_dydp<<<grid, nShells, sizeof(double)*nShells>>> (deviceInput.lstack_rlm, deviceInput.idx_gl_1d_rlm_j, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.p_jl, deviceInput.asin_theta_1d_rtm, constants);
  cudaDevSync();
  transBwdVec_dy_dp.endTimer(); 

  static Timer transBwdScalar("bwd scalar transform");
  cudaPerformance.registerTimer(&transBwdScalar);
  transBwdScalar.startTimer(); 
  /*transB_scalar_reduction<32,
                        hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
                      double>
                <<<grid, 32>>> (deviceInput.lstack_rlm, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.p_jl, constants);*/
  transB_scalar<<<grid, nShells>>> (deviceInput.lstack_rlm, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.p_jl, constants);
  cudaDevSync();
  transBwdScalar.endTimer(); 
}
