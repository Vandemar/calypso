#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>
#include <unistd.h>

Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;

int countFT=0, countBT=0;

hipStream_t streams[2];
//__constant__ Geometry_c devConstants;

void initialize_gpu_() {
  int device_count, device;
  // Gets number of GPU devices
  hipGetDeviceCount(&device_count);
  hipGetDevice(&device);
  hipDeviceReset();
  #if defined(CUDA_TIMINGS)
    hipProfilerStart();
  #endif
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  hipFree(0);
}

void set_constants_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *trunc_lvl, int *np_smp) {

  for(int i=0; i<3; i++) { 
    constants.nidx_rtm[i] = nidx_rtm[i];
    constants.istep_rtm[i] = istep_rtm[i];
  }

  for(int i=0; i<2; i++) {
    constants.nidx_rlm[i] = nidx_rlm[i];
    constants.istep_rlm[i] = istep_rlm[i];
  }

  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.t_lvl = *trunc_lvl; 

  constants.np_smp = *np_smp;

  #if defined(CUDA_OTF)
    initDevConstVariables();
  #endif

  for(unsigned int i=0; i<2; i++)       
    cudaErrorCheck(hipStreamCreate(&streams[i]));


//  #if defined(CUDA_TIMINGS)
/*    t_1 = MPI_Wtime();
    char name[15];
    gethostname(name, 15);
    string str(name);
    std::cout<<"Host: " << str << "\t Memory Allocation Time: " << t_1-t_0 << "\t Device Initialization Time: " << t_3-t_2 << std::endl;*/
//  #endif

}

void setptrs_(int *idx_gl_1d_rlm_j) {
  //Necessary to filter harmonic modes across MPI nodes.
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
}

/*void setptrs_(int *idx_gl_1d_rlm_j, double *P_smdt, double *dP_smdt) {
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
  //h_debug.P_smdt = P_smdt;
  //h_debug.dP_smdt = dP_smdt;
}*/


void initialize_leg_trans_gpu_() {
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_colat_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.asin_theta_1d_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.lstack_rlm), (constants.nidx_rtm[2]+1)*sizeof(int))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rlm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm_7), constants.nidx_rlm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.idx_gl_1d_rlm_j), constants.nidx_rlm[1]*3*sizeof(int))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.radius_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.weight_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_p_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_n_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF) || defined(CUDA_OTF)
    h_debug.P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    h_debug.dP_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    #ifdef CUDA_OTF
    cudaErrorCheck(hipMalloc((void**)&(d_debug.P_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    cudaErrorCheck(hipMemset(d_debug.P_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    cudaErrorCheck(hipMalloc((void**)&(d_debug.dP_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    cudaErrorCheck(hipMemset(d_debug.dP_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    #endif
  #endif
}
 
void alloc_space_on_gpu_(int *ncmp, int *nvector, int *nscalar) {
  int ncomp = constants.ncomp = *ncmp;
  constants.nvector = *nvector;
  constants.nscalar = *nscalar;

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    if(!h_debug.vr_rtm)
      h_debug.vr_rtm = (double*) malloc (sizeof(double)*constants.nnod_rtm*constants.ncomp);
    if(!h_debug.sp_rlm)
      h_debug.sp_rlm = (double*) malloc (sizeof(double)*constants.nnod_rlm*constants.ncomp);
  #endif

  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  if(!deviceInput.vr_rtm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  }
  if(!deviceInput.sp_rlm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
  }
}

void memcpy_h2d_(int *lstack_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *g_sph_rlm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, int *idx_gl_1d_rlm_j, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm) {
    h_debug.lstack_rlm = lstack_rlm;
 #ifdef CUDA_DEBUG 
    h_debug.g_colat_rtm = g_colat_rtm;
    h_debug.g_sph_rlm = g_sph_rlm;
#endif

  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.asin_theta_1d_rtm, asin_theta_1d_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.lstack_rlm, lstack_rlm, (constants.nidx_rtm[2]+1)*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm_7, g_sph_rlm_7, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.idx_gl_1d_rlm_j, idx_gl_1d_rlm_j, constants.nidx_rlm[1]*3*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.radius_1d_rlm_r, radius_1d_rlm_r, constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.weight_rtm, weight_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_p_rlm_rtm, mdx_p_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_n_rlm_rtm, mdx_n_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 
}

void cpy_schmidt_2_gpu_(double *P_jl, double *dP_jl, double *P_rtm, double *dP_rtm) {
    cudaErrorCheck(hipMemcpy(deviceInput.p_jl, P_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_jl, dP_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.p_rtm, P_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_rtm, dP_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
}
 
void cpy_field_dev2host_4_debug_() {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void cpy_spec_dev2host_4_debug_() {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void set_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
}

void set_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
}

void retrieve_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
}

void clear_spectrum_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*(*ncomp)*sizeof(double)));
}

void clear_field_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*(*ncomp)*sizeof(double)));
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm_7));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.lstack_rlm));
    cudaErrorCheck(hipFree(deviceInput.idx_gl_1d_rlm_j));
    cudaErrorCheck(hipFree(deviceInput.radius_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.weight_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_p_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_n_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.asin_theta_1d_rtm));
    cudaErrorCheck(hipFree(deviceInput.p_jl));
    cudaErrorCheck(hipFree(deviceInput.dP_jl));
    cudaErrorCheck(hipFree(deviceInput.p_rtm));
    cudaErrorCheck(hipFree(deviceInput.dP_rtm));
}

void deAllocDebugMem() {
  #if defined(CUDA_OTF) 
    free(h_debug.P_smdt);
    free(h_debug.dP_smdt);
  #endif
    free(h_debug.vr_rtm);
    free(h_debug.sp_rlm);
//  free(h_debug.g_sph_rlm);
  #if defined(CUDA_OTF) 
    cudaErrorCheck(hipFree(d_debug.P_smdt));
    cudaErrorCheck(hipFree(d_debug.dP_smdt));
  #endif
//  cudaErrorCheck(hipFree(d_debug.g_sph_rlm));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
  for(int i=0; i<2; i++)
    cudaErrorCheck(hipStreamDestroy(streams[i]));
  #if defined(CUDA_TIMINGS)
    hipProfilerStop();
  #endif
}

void cuda_sync_device_() {
  cudaErrorCheck(hipDeviceSynchronize());
}

void initDevConstVariables() {
  hipError_t error;
  error = hipMemcpyToSymbol(HIP_SYMBOL(devConstants), &constants, sizeof(Geometry_c), 0, hipMemcpyHostToDevice);
  cudaErrorCheck(error);
}
