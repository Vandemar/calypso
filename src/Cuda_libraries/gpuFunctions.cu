#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <unistd.h>

#include "legendre_poly.h"

#include "math_functions.h"
#include "hip/hip_math_constants.h"

hipDeviceProp_t prop;
Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;
References hostData;
Logger cudaPerformance("Metrics.log", 7);

Timer movData2GPU;
Timer movData2Host;

int countFT=0, countBT=0;
int minGridSize=0, blockSize=0;
size_t devMemory = 0;
hipStream_t *streams;
int nStreams=0;

// **** lstack_rlm resides in global memory as well as constant memory
// ** Pick one or the other
__constant__ int lstack_rlm_cmem[1000];

//CUDA Unbound - part of device reduce example
bool g_verbose = false; // Whether to display input/output to console
hipcub::CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

void initialize_gpu_() {

//Required because, Template parameters need to be evaluated by compile time
//#if __cplusplus > 199711L
//   #error c++ 11 standard or greater REQUIRED!
// #endif

  int device_count, device;
  hipDeviceReset();
  // Gets number of GPU devices
  hipGetDeviceCount(&device_count);
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  devMemory = prop.totalGlobalMem;
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
  cudaErrorCheck(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
  hipFree(0);
  #if defined(CUDA_TIMINGS)
    hipProfilerStart();
  #endif
  //registerAllTimers();
  movData2GPU.setWhatAmI("Transfer data from Host to GPU");
  movData2Host.setWhatAmI("Transfer data from GPU to Host");

  cudaPerformance.registerTimer(&movData2GPU);
  cudaPerformance.registerTimer(&movData2Host);
}

void registerAllTimers() {
  //If more timers are registered than the amount specified in the constructor of logger, program will 
  // segfault.
  // TO BE DEPRECATED
  // Timer transBwdVec("Bwd Vector Transform");
  // Timer transBwdScalar("Bwd Scalar Transform");
  // Timer transF_s("Fwd scalar transform with cached schmidt");
  // Timer transF_reduce("Fwd Vector Reduction Algorithm");
  // cudaPerformance.registerTimer(&transBwdVec);
  // cudaPerformance.registerTimer(&transBwdScalar);
  // cudaPerformance.registerTimer(&transF_reduce);
  // cudaPerformance.registerTimer(&transF_s);
} 

void set_constants_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *trunc_lvl, int *np_smp) {

  cudaPerformance.recordProblemDescription(*trunc_lvl, nidx_rtm[0], nidx_rtm[1]);

  //For best occupancy
  /*cudaErrorCheck(hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize,
																&blockSize,
																transF_vec_reduction< 10, 
																  hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
																		double>, 
																computeSharedMemory));  
  */

  for(int i=0; i<3; i++) { 
    constants.nidx_rtm[i] = nidx_rtm[i];
    constants.istep_rtm[i] = istep_rtm[i];
  }

  for(int i=0; i<2; i++) {
    constants.nidx_rlm[i] = nidx_rlm[i];
    constants.istep_rlm[i] = istep_rlm[i];
  }

  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.t_lvl = *trunc_lvl; 

  constants.np_smp = *np_smp;



//  #if defined(CUDA_TIMINGS)
/*    t_1 = MPI_Wtime();
    char name[15];
    gethostname(name, 15);
    string str(name);
    std::cout<<"Host: " << str << "\t Memory Allocation Time: " << t_1-t_0 << "\t Device Initialization Time: " << t_3-t_2 << std::endl;*/
//  #endif

}

void setptrs_(int *idx_gl_1d_rlm_j) {
  //Necessary to filter harmonic modes across MPI nodes.
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
}

/*void setptrs_(int *idx_gl_1d_rlm_j, double *P_smdt, double *dP_smdt) {
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
  //h_debug.P_smdt = P_smdt;
  //h_debug.dP_smdt = dP_smdt;
}*/


void initialize_leg_trans_gpu_() {
  size_t memAllocation = 0;
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_colat_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[0]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.asin_theta_1d_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.lstack_rlm), (constants.nidx_rtm[2]+1)*sizeof(int))); 
  
  memAllocation -= (constants.nidx_rtm[2]+1)*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rlm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm_7), constants.nidx_rlm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.idx_gl_1d_rlm_j), constants.nidx_rlm[1]*3*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*3*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.radius_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[0]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.weight_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_p_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_n_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(int);
//#ifndef CUDA_OTF
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
//#endif
//OTF has yet to be implemented for fwd transform
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);

// Question, is loading from DRAM faster than actual calculation? 
//since m=0,l=0 is the trivial case, this is excluded. All others i.e, m=1 upto t_lvl (inclusive) is allocated 
//  cudaErrorCheck(hipMalloc((void**)&(deviceInput.leg_poly_m_eq_l), sizeof(double)*(constants.t_lvl)));
//  memAllocation += sizeof(double)*(constants.t_lvl);

// A variable amount of memory
  // dim3 grid(1,1,1);
  // dim3 block(64,1,1);
  // set_leg_poly_m_ep_l<<<grid,block,0>>>(deviceInput.leg_poly_m_eq_l);
  
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    h_debug.P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    h_debug.dP_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    cudaErrorCheck(hipMalloc((void**)&(d_debug.P_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    memAllocation -= sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1];
    cudaErrorCheck(hipMemset(d_debug.P_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    cudaErrorCheck(hipMalloc((void**)&(d_debug.dP_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    memAllocation -= sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1];
    cudaErrorCheck(hipMemset(d_debug.dP_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  #endif

  unsigned int numberOfDoubles = memAllocation/(sizeof(double));
  unsigned int numberOfReductionSpaces = min(numberOfDoubles/(constants.nidx_rtm[1]*3), constants.nidx_rtm[1]);
  //streams = (hipStream_t*) malloc (sizeof(hipStream_t) * numberOfReductionSpaces);
  streams = new hipStream_t[numberOfReductionSpaces];
  for(int i=0; i<numberOfReductionSpaces; i++) {
    cudaErrorCheck(hipStreamCreate(&streams[i]));
    nStreams++;
  }
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.reductionSpace), sizeof(double)*numberOfReductionSpaces*3*constants.nidx_rtm[1]));
  memAllocation -= sizeof(double)*numberOfReductionSpaces*3*constants.nidx_rtm[1];
  if(memAllocation <= 0) {
    exit(-1);
  }
}
 
void alloc_space_on_gpu_(int *ncmp, int *nvector, int *nscalar) {
  int ncomp = constants.ncomp = *ncmp;
  constants.nvector = *nvector;
  constants.nscalar = *nscalar;

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    if(!h_debug.vr_rtm)
      h_debug.vr_rtm = (double*) malloc (sizeof(double)*constants.nnod_rtm*constants.ncomp);
    if(!h_debug.sp_rlm)
      h_debug.sp_rlm = (double*) malloc (sizeof(double)*constants.nnod_rlm*constants.ncomp);
  #endif

  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  if(!deviceInput.vr_rtm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  }
  if(!deviceInput.sp_rlm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
  }
}

void memcpy_h2d_(int *lstack_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *g_sph_rlm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, int *idx_gl_1d_rlm_j, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm) {
   
    hostData.mdx_p_rlm_rtm = mdx_p_rlm_rtm;
    hostData.mdx_n_rlm_rtm = mdx_n_rlm_rtm;
    hostData.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
    hostData.radius_1d_rlm_r = radius_1d_rlm_r;
    hostData.g_sph_rlm_7= g_sph_rlm_7;

    h_debug.lstack_rlm = lstack_rlm;
 #ifdef CUDA_DEBUG 
    h_debug.g_colat_rtm = g_colat_rtm;
    h_debug.g_sph_rlm = g_sph_rlm;
 #endif

  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.asin_theta_1d_rtm, asin_theta_1d_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.lstack_rlm, lstack_rlm, (constants.nidx_rtm[2]+1)*sizeof(int), hipMemcpyHostToDevice)); 
 cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(lstack_rlm_cmem), lstack_rlm, sizeof(int) * (constants.nidx_rtm[2]+1), 0, hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm_7, g_sph_rlm_7, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.idx_gl_1d_rlm_j, idx_gl_1d_rlm_j, constants.nidx_rlm[1]*3*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.radius_1d_rlm_r, radius_1d_rlm_r, constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.weight_rtm, weight_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_p_rlm_rtm, mdx_p_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_n_rlm_rtm, mdx_n_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 
}

void cpy_schmidt_2_gpu_(double *P_jl, double *dP_jl, double *P_rtm, double *dP_rtm) {
  //#ifndef CUDA_OTF
    cudaErrorCheck(hipMemcpy(deviceInput.p_jl, P_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_jl, dP_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
  //#endif
//FWD trans OTF has yet to be implemented
    cudaErrorCheck(hipMemcpy(deviceInput.p_rtm, P_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_rtm, dP_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
}
 
void cpy_field_dev2host_4_debug_() {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void cpy_spec_dev2host_4_debug_() {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void set_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  movData2GPU.startTimer();
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
  movData2GPU.endTimer();
}

void set_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  movData2GPU.startTimer();
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
  movData2GPU.endTimer();
}

void retrieve_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  movData2Host.startTimer();
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
  movData2Host.endTimer();
}

void retrieve_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  movData2Host.startTimer();
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
  movData2Host.endTimer();
}

//How should these functions be timed?
void clear_spectrum_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*(*ncomp)*sizeof(double)));
}

void clear_field_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*(*ncomp)*sizeof(double)));
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm_7));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.lstack_rlm));
    cudaErrorCheck(hipFree(deviceInput.idx_gl_1d_rlm_j));
    cudaErrorCheck(hipFree(deviceInput.radius_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.weight_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_p_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_n_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.asin_theta_1d_rtm));
  #ifndef CUDA_OTF
    cudaErrorCheck(hipFree(deviceInput.p_jl));
    cudaErrorCheck(hipFree(deviceInput.dP_jl));
    cudaErrorCheck(hipFree(deviceInput.p_rtm));
    cudaErrorCheck(hipFree(deviceInput.dP_rtm));
  #endif
    cudaErrorCheck(hipFree(deviceInput.reductionSpace));    
}

void deAllocDebugMem() {
  #if defined(CUDA_OTF) 
    free(h_debug.P_smdt);
    free(h_debug.dP_smdt);
  #endif
    free(h_debug.vr_rtm);
    free(h_debug.sp_rlm);
//  free(h_debug.g_sph_rlm);
  #if defined(CUDA_OTF) 
    cudaErrorCheck(hipFree(d_debug.P_smdt));
    cudaErrorCheck(hipFree(d_debug.dP_smdt));
  #endif
//  cudaErrorCheck(hipFree(d_debug.g_sph_rlm));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
  for(int i=0; i<nStreams; i++)
    cudaErrorCheck(hipStreamDestroy(streams[i]));
  #if defined(CUDA_TIMINGS)
    hipProfilerStop();
  #endif

  //Write performance metrics
  cudaPerformance.echoAllClocks();
  cudaPerformance.closeStream();
}

//Fortran wrapper function
void cuda_sync_device_() {
  cudaErrorCheck(hipDeviceSynchronize());
}

void cudaDevSync() {
  cudaErrorCheck(hipDeviceSynchronize());
}

size_t computeSharedMemory(int blockSize) {
  return blockSize * sizeof(double);
} 
