#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <unistd.h>

#include "legendre_poly.h"

#include "math_functions.h"
#include "hip/hip_math_constants.h"

hipDeviceProp_t prop;
Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;
References hostData;

symmetricModes *pairedModes;
unsymmetricModes *unpairedModes;

#ifdef CUDA_TIMINGS
  Logger cudaPerformance("Metrics.log", 7);

Timer movData2GPU;
Timer movData2Host;
#endif

int countFT=0, countBT=0;
int minGridSize=0, blockSize=0;
size_t devMemory = 0;
hipStream_t *streams;
int nStreams=0;

#ifdef CUBLAS
hipblasHandle_t handle;
hipblasStatus_t statusCublas;
deviceBUFFERS fwdTransBuf;
#endif

// **** lstack_rlm resides in global memory as well as constant memory
// ** Pick one or the other
//__constant__ int lstack_rlm_cmem[1000];

//CUDA Unbound - part of device reduce example
//bool g_verbose = false; // Whether to display input/output to console
//hipcub::CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

void initialize_gpu_() {

//Required because, Template parameters need to be evaluated by compile time
//#if __cplusplus > 199711L
//   #error c++ 11 standard or greater REQUIRED!
// #endif

  int device_count, device;
  // Gets number of GPU devices
  hipGetDeviceCount(&device_count);
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  devMemory = prop.totalGlobalMem;
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
//  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  cudaErrorCheck(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
  hipFree(0);
  #if defined(CUDA_TIMINGS)
    hipProfilerStart();

  //registerAllTimers();
  movData2GPU.setWhatAmI("Transfer data from Host to GPU");
  movData2Host.setWhatAmI("Transfer data from GPU to Host");

  cudaPerformance.registerTimer(&movData2GPU);
  cudaPerformance.registerTimer(&movData2Host);
  #endif

  #ifdef CUBLAS
    cublasStatusCheck(hipblasCreate(&handle));
  #endif
}

void registerAllTimers() {
  //If more timers are registered than the amount specified in the constructor of logger, program will 
  // segfault.
  // TO BE DEPRECATED
  // Timer transBwdVec("Bwd Vector Transform");
  // Timer transBwdScalar("Bwd Scalar Transform");
  // Timer transF_s("Fwd scalar transform with cached schmidt");
  // Timer transF_reduce("Fwd Vector Reduction Algorithm");
  // cudaPerformance.registerTimer(&transBwdVec);
  // cudaPerformance.registerTimer(&transBwdScalar);
  // cudaPerformance.registerTimer(&transF_reduce);
  // cudaPerformance.registerTimer(&transF_s);
} 

void set_constants_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *trunc_lvl, int *np_smp) {

#if defined(CUDA_TIMINGS)
  cudaPerformance.recordProblemDescription(*trunc_lvl, nidx_rtm[0], nidx_rtm[1]);
#endif

  //For best occupancy
  /*cudaErrorCheck(hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize,
																&blockSize,
																transF_vec_reduction< 10, 
																  hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,
																		double>, 
																computeSharedMemory));  
  */

  for(int i=0; i<3; i++) { 
    constants.nidx_rtm[i] = nidx_rtm[i];
    constants.istep_rtm[i] = istep_rtm[i];
  }

  for(int i=0; i<2; i++) {
    constants.nidx_rlm[i] = nidx_rlm[i];
    constants.istep_rlm[i] = istep_rlm[i];
  }

  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.t_lvl = *trunc_lvl; 

  constants.np_smp = *np_smp;



//  #if defined(CUDA_TIMINGS)
/*    t_1 = MPI_Wtime();
    char name[15];
    gethostname(name, 15);
    string str(name);
    std::cout<<"Host: " << str << "\t Memory Allocation Time: " << t_1-t_0 << "\t Device Initialization Time: " << t_3-t_2 << std::endl;*/
//  #endif

}

void setptrs_(int *idx_gl_1d_rlm_j) {
  //Necessary to filter harmonic modes across MPI nodes.
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
}

/*void setptrs_(int *idx_gl_1d_rlm_j, double *P_smdt, double *dP_smdt) {
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
  //h_debug.P_smdt = P_smdt;
  //h_debug.dP_smdt = dP_smdt;
}*/


void initialize_leg_trans_gpu_() {
  size_t memAllocation = 0;
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_colat_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[0]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.asin_theta_1d_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.lstack_rlm), (constants.nidx_rtm[2]+1)*sizeof(int))); 
  
  memAllocation -= (constants.nidx_rtm[2]+1)*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rlm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm_7), constants.nidx_rlm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(double);
  
  
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.idx_gl_1d_rlm_j), constants.nidx_rlm[1]*3*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*3*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.radius_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[0]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.weight_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  memAllocation -= constants.nidx_rtm[1]*sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_p_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.mdx_n_rlm_rtm), constants.nidx_rlm[1]*sizeof(int))); 
  memAllocation -= constants.nidx_rlm[1]*sizeof(int);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.p_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_rtm), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  memAllocation -= constants.nidx_rtm[1]*constants.nidx_rlm[1] * sizeof(double);
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.Pgvw), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  
// Question, is loading from DRAM faster than actual calculation? 
//since m=0,l=0 is the trivial case, this is excluded. All others i.e, m=1 upto t_lvl (inclusive) is allocated 
//  cudaErrorCheck(hipMalloc((void**)&(deviceInput.leg_poly_m_eq_l), sizeof(double)*(constants.t_lvl)));
//  memAllocation += sizeof(double)*(constants.t_lvl);

// A variable amount of memory
  // dim3 grid(1,1,1);
  // dim3 block(64,1,1);
  // set_leg_poly_m_ep_l<<<grid,block,0>>>(deviceInput.leg_poly_m_eq_l);
  
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    h_debug.P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    h_debug.dP_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
    cudaErrorCheck(hipMalloc((void**)&(d_debug.P_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    memAllocation -= sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1];
    cudaErrorCheck(hipMemset(d_debug.P_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    cudaErrorCheck(hipMalloc((void**)&(d_debug.dP_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
    memAllocation -= sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1];
    cudaErrorCheck(hipMemset(d_debug.dP_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  #endif

  unsigned int numberOfDoubles = memAllocation/(sizeof(double));
  unsigned int numberOfReductionSpaces = min(numberOfDoubles/(constants.nidx_rtm[1]*3), constants.nidx_rtm[1]);
  //streams = (hipStream_t*) malloc (sizeof(hipStream_t) * numberOfReductionSpaces)
  numberOfReductionSpaces=32;
  streams = new hipStream_t[numberOfReductionSpaces];
  for(int i=0; i<numberOfReductionSpaces; i++) {
    cudaErrorCheck(hipStreamCreate(&streams[i]));
    nStreams++;
  }
//  cudaErrorCheck(hipMalloc((void**)&(deviceInput.reductionSpace), sizeof(double)*numberOfReductionSpaces*3*constants.nidx_rtm[1]));
//  memAllocation -= sizeof(double)*numberOfReductionSpaces*3*constants.nidx_rtm[1];
//  if(memAllocation <= 0) {
//    exit(-1);
//  }

#ifdef CUBLAS
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.d_vr_p_0), sizeof(double) * (constants.nvector) * constants.nidx_rtm[0] * constants.nidx_rtm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.d_vr_p_1), sizeof(double) * (constants.nvector) * constants.nidx_rtm[0] * constants.nidx_rtm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.d_vr_p_2), sizeof(double) * (constants.nvector) * constants.nidx_rtm[0] * constants.nidx_rtm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.d_vr_n_0), sizeof(double) * (constants.nvector) * constants.nidx_rtm[0] * constants.nidx_rtm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.d_vr_n_1), sizeof(double) * (constants.nvector) * constants.nidx_rtm[0] * constants.nidx_rtm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.pol_e), sizeof(double) * (constants.nvector) * constants.nidx_rlm[0] * constants.nidx_rlm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.dpoldt_e), sizeof(double) * (constants.nvector) * constants.nidx_rlm[0] * constants.nidx_rlm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.dpoldp_e), sizeof(double) * (constants.nvector) * constants.nidx_rlm[0] * constants.nidx_rlm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.dtordt_e), sizeof(double) * (constants.nvector) * constants.nidx_rlm[0] * constants.nidx_rlm[1])); 
  cudaErrorCheck(hipMalloc((void**)&(fwdTransBuf.dtordp_e), sizeof(double) * (constants.nvector) * constants.nidx_rlm[0] * constants.nidx_rlm[1])); 
#endif
}
 
void alloc_space_on_gpu_(int *ncmp, int *nvector, int *nscalar) {
  int ncomp = constants.ncomp = *ncmp;
  constants.nvector = *nvector;
  constants.nscalar = *nscalar;

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    if(!h_debug.vr_rtm)
      h_debug.vr_rtm = (double*) malloc (sizeof(double)*constants.nnod_rtm*constants.ncomp);
    if(!h_debug.sp_rlm)
      h_debug.sp_rlm = (double*) malloc (sizeof(double)*constants.nnod_rlm*constants.ncomp);
  #endif

  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  if(!deviceInput.vr_rtm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  }
  if(!deviceInput.sp_rlm) {
    cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
    cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
  }
}

void memcpy_h2d_(int *lstack_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *g_sph_rlm, double *g_sph_rlm_7, double *asin_theta_1d_rtm, int *idx_gl_1d_rlm_j, double *radius_1d_rlm_r, double *weight_rtm, int *mdx_p_rlm_rtm, int *mdx_n_rlm_rtm) {
   
    hostData.mdx_p_rlm_rtm = mdx_p_rlm_rtm;
    hostData.mdx_n_rlm_rtm = mdx_n_rlm_rtm;
    hostData.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
    hostData.radius_1d_rlm_r = radius_1d_rlm_r;
    hostData.g_sph_rlm_7= g_sph_rlm_7;

    h_debug.lstack_rlm = lstack_rlm;
 #ifdef CUDA_DEBUG 
    h_debug.g_colat_rtm = g_colat_rtm;
    h_debug.g_sph_rlm = g_sph_rlm;
 #endif

  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.asin_theta_1d_rtm, asin_theta_1d_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.lstack_rlm, lstack_rlm, (constants.nidx_rtm[2]+1)*sizeof(int), hipMemcpyHostToDevice)); 
 cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(lstack_rlm_cmem), lstack_rlm, sizeof(int) * (constants.nidx_rtm[2]+1), 0, hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm_7, g_sph_rlm_7, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  
  findSymmetricModes(idx_gl_1d_rlm_j);
  cudaErrorCheck(hipMalloc((void**)&deviceInput.pairedList, constants.nPairs * sizeof(symmetricModes))); 
  cudaErrorCheck(hipMalloc((void**)&deviceInput.unpairedList, constants.nSingletons * sizeof(unsymmetricModes)));
  cudaErrorCheck(hipMemcpy(deviceInput.pairedList, pairedModes, constants.nPairs * sizeof(symmetricModes), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.unpairedList, unpairedModes, constants.nSingletons * sizeof(unsymmetricModes), hipMemcpyHostToDevice));

  cudaErrorCheck(hipMemcpy(deviceInput.idx_gl_1d_rlm_j, idx_gl_1d_rlm_j, constants.nidx_rlm[1]*3*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.radius_1d_rlm_r, radius_1d_rlm_r, constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.weight_rtm, weight_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_p_rlm_rtm, mdx_p_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.mdx_n_rlm_rtm, mdx_n_rlm_rtm, constants.nidx_rlm[1]*sizeof(int), hipMemcpyHostToDevice)); 

 
  //cpy_schidt_2_gpu_ has already been executed at this point 
#ifdef CUBLAS
  normalizeLegendre<<<constants.nidx_rlm[1], constants.nidx_rtm[1], 0, streams[0]>>>(deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.Pgvw, deviceInput.g_sph_rlm_7, deviceInput.weight_rtm, deviceInput.asin_theta_1d_rtm, deviceInput.idx_gl_1d_rlm_j, constants); 
#else
  normalizeLegendre<<<constants.nidx_rlm[1], constants.nidx_rtm[1], 0, streams[0]>>>(deviceInput.p_rtm, deviceInput.dP_rtm, deviceInput.g_sph_rlm_7, deviceInput.weight_rtm, constants); 
#endif

}

void cpy_schmidt_2_gpu_(double *P_jl, double *dP_jl, double *P_rtm, double *dP_rtm) {
  //#ifndef CUDA_OTF
    cudaErrorCheck(hipMemcpy(deviceInput.p_jl, P_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_jl, dP_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
  //#endif
//FWD trans OTF has yet to be implemented
    cudaErrorCheck(hipMemcpy(deviceInput.p_rtm, P_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceInput.dP_rtm, dP_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
  #ifdef CUBLAS
    cudaErrorCheck(hipMemcpy(deviceInput.Pgvw, P_rtm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyHostToDevice));
  #endif


}


void cpy_field_dev2host_4_debug_(int *ncomp) {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void cpy_spec_dev2host_4_debug_(int *ncomp) {
  #if defined(CUDA_OTF)
    cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  #endif
  cudaErrorCheck(hipMemcpy(h_debug.sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void set_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
}

void set_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyHostToDevice)); 
}

void retrieve_spectrum_data_(double *sp_rlm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_spectrum_data_cuda_and_org_(double *sp_rlm, int *ncomp, int *kst, int *ked) {
  int idx = (*ncomp) * ((*kst)-1) * constants.istep_rlm[0];
  int maxIdx = (*ncomp) + (*ncomp) * ((*ked-1) * constants.istep_rlm[0] + constants.istep_rlm[1]*(constants.nidx_rlm[1]-1));
  cudaErrorCheck(hipMemcpy(&sp_rlm[idx], &deviceInput.sp_rlm[idx], (maxIdx - idx)*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_cuda_and_org_(double *vr_rtm, int *ncomp, int *mStart, int *mEnd) {
  int idx = (*ncomp) * ((*mStart - 1) * constants.istep_rtm[2]);
  int maxIdx = (*ncomp) + (*ncomp) * ((constants.nidx_rtm[1]-1)*constants.istep_rtm[1] + (constants.nidx_rtm[0]-1)*constants.istep_rtm[0] +  (*mEnd-1)*constants.istep_rtm[2]); 
  cudaErrorCheck(hipMemcpy(&vr_rtm[idx], &deviceInput.vr_rtm[idx], (maxIdx - idx)*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_(double *vr_rtm, int *ncomp) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*(*ncomp)*sizeof(double), hipMemcpyDeviceToHost)); 
}

//How should these functions be timed?
void clear_spectrum_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*(*ncomp)*sizeof(double)));
}

void clear_field_data_(int *ncomp) {
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*(*ncomp)*sizeof(double)));
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm_7));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.lstack_rlm));
    cudaErrorCheck(hipFree(deviceInput.idx_gl_1d_rlm_j));
    cudaErrorCheck(hipFree(deviceInput.radius_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.weight_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_p_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.mdx_n_rlm_rtm));
    cudaErrorCheck(hipFree(deviceInput.asin_theta_1d_rtm));
  #ifndef CUDA_OTF
    cudaErrorCheck(hipFree(deviceInput.p_jl));
    cudaErrorCheck(hipFree(deviceInput.dP_jl));
    cudaErrorCheck(hipFree(deviceInput.p_rtm));
    cudaErrorCheck(hipFree(deviceInput.dP_rtm));
  #endif
    cudaErrorCheck(hipFree(deviceInput.reductionSpace));    
}

void deAllocDebugMem() {
  #if defined(CUDA_OTF) 
    free(h_debug.P_smdt);
    free(h_debug.dP_smdt);
  #endif
    free(h_debug.vr_rtm);
    free(h_debug.sp_rlm);
//  free(h_debug.g_sph_rlm);
  #if defined(CUDA_OTF) 
    cudaErrorCheck(hipFree(d_debug.P_smdt));
    cudaErrorCheck(hipFree(d_debug.dP_smdt));
  #endif
//  cudaErrorCheck(hipFree(d_debug.g_sph_rlm));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
  for(int i=0; i<nStreams; i++)
    cudaErrorCheck(hipStreamDestroy(streams[i]));
  #if defined(CUDA_TIMINGS)
    hipProfilerStop();
  #endif

  hipDeviceReset();

#if defined(CUDA_TIMINGS)
  //Write performance metrics
  cudaPerformance.echoAllClocks();
  cudaPerformance.closeStream();
#endif
}

//Fortran wrapper function
void cuda_sync_device_() {
  cudaErrorCheck(hipDeviceSynchronize());
}

void cudaDevSync() {
  cudaErrorCheck(hipDeviceSynchronize());
}

size_t computeSharedMemory(int blockSize) {
  return blockSize * sizeof(double);
} 

int searchMode(int *idx_j, int order, int degree) {
  for(int i=0; i<constants.nidx_rlm[1]; i++) {
    if(idx_j[constants.nidx_rlm[1]*2+i] == order && idx_j[constants.nidx_rlm[1]+i] == degree)
      return i;
  }
  return -1;
}

//Note that the sign of the order is not preserved for pairs of modes.**
void findSymmetricModes(int *idx_gl_1d_rlm_j) {
  //At most there are nModes/2 pairs.
  //At most there are nModes of no pairs. 

  symmetricModes *pmTmp = new symmetricModes[constants.nidx_rlm[1]/2];
  unsymmetricModes *upmTmp = new unsymmetricModes[constants.nidx_rlm[1]]; 

  int nPM=0, nUPM=0;

  //Order=0
  for(int l=0; l<=constants.t_lvl; l++) {
    int idx = searchMode(idx_gl_1d_rlm_j, 0, l);
    if(idx != -1) {
      upmTmp[nUPM].modeIdx = idx;
      upmTmp[nUPM].order = 0;
      nUPM++;
    }
  }
  
  //Order >= 1  
  for(int m=1; m<=constants.t_lvl; m++) {
    int idxP, idxN;
    for(int l=m; l<=constants.t_lvl; l++) {
      idxP = searchMode(idx_gl_1d_rlm_j, m, l); 
      idxN = searchMode(idx_gl_1d_rlm_j, -1*m, l); 
      if(idxP != -1 && idxN != -1) {
        pmTmp[nPM].positiveModeIdx = idxP;
        pmTmp[nPM].negativeModeIdx = idxN;
        pmTmp[nPM].order = m;
        nPM++;
      } else if (idxP != -1) {
        upmTmp[nUPM].modeIdx = idxP;
        upmTmp[nUPM].order = m;
        nUPM++;
      } else if (idxN != -1) {
        upmTmp[nUPM].modeIdx = idxN;
        upmTmp[nUPM].order = -1*m;
        nUPM++;
      } else
        continue;
    }
  }

  pairedModes = new symmetricModes[nPM];
  unpairedModes = new unsymmetricModes[nUPM];

  for(int i=0; i<nPM; i++) {
    pairedModes[i].positiveModeIdx = pmTmp[i].positiveModeIdx;
    pairedModes[i].negativeModeIdx = pmTmp[i].negativeModeIdx;
    pairedModes[i].order = pmTmp[i].order;
  }

  for(int i=0; i<nUPM; i++) {
    unpairedModes[i].modeIdx = upmTmp[i].modeIdx;
    unpairedModes[i].order = upmTmp[i].order;
  }

  constants.nPairs = nPM;  
  constants.nSingletons = nUPM;

  delete [] pmTmp;  
  delete [] upmTmp;  
}
