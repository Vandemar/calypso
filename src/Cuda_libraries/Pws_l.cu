#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"


extern "C" void spectral_to_grid_(int* nidx_rlm, int* nidx_rtm, double* P_rtm, double* g_sph_rlm, double* weight_rtm, double* Pws) {
  
  int i;
  //device matrices 
  double *P_rtm_d, *g_sph_rlm_d, *weight_rtm_d;
  hipblasHandle_t handle;
        
  //allocating space for device matrices
  hipMalloc((void**)&P_rtm_d, sizeof(*P_rtm)*(*nidx_rtm)*(*nidx_rlm));
 
  hipblasSetMatrix(*nidx_rtm, *nidx_rlm, sizeof(*P_rtm), P_rtm, *nidx_rtm, P_rtm_d, *nidx_rtm);
  
  hipblasCreate(&handle);
  double *tmp;
  for( i=1; i<(*nidx_rlm)+1; i++) {
    tmp = g_sph_rlm+(i-1);
    hipblasDscal(handle, *nidx_rtm, tmp, P_rtm_d + (*nidx_rtm)*(i-1), 1); 
  } 

  for( i=1; i<(*nidx_rtm)+1; i++) {
    tmp = weight_rtm+(i-1);
    hipblasDscal(handle, *nidx_rlm, tmp, P_rtm_d + (i-1), *nidx_rtm);    
  }

  //moving device data into a host matrix
  hipblasGetMatrix(*nidx_rtm, *nidx_rlm, sizeof(*Pws), P_rtm_d, *nidx_rtm, Pws, *nidx_rtm);   
  
  hipblasDestroy(handle);

  hipFree(P_rtm_d);
  return; 
} 
