#include "legendre_poly.h"

//Helper functions are defined.
/*
template<class T>
void cudaErrorCheck(T error) {
  if ( strcmp(_cudaGetErrorEnum(error), "hipSuccess") != 0 ) {
    printf ("%s\n", _cudaGetErrorEnum(error));
  }
  return;
}
*/

void cudaErrorCheck(hipError_t error) {
  if ( strcmp(_cudaGetErrorEnum(error), "hipSuccess") != 0 ) {
    printf ("%s\n", _cudaGetErrorEnum(error));
  }
  return;
}

void cudaErrorCheck(hipfftResult error) {
  if ( strcmp(_cudaGetErrorEnum(error), "hipSuccess") != 0 ) {
    printf ("%s\n", _cudaGetErrorEnum(error));
  }
  return;
}

void cublasStatusCheck(hipblasStatus_t stat) {
  if ( strcmp(_cublasGetErrorEnum(stat),"HIPBLAS_STATUS_SUCCESS") != 0 ) {
    printf("%s\n", _cublasGetErrorEnum(stat));
  }
  return;
}

int findMin(int *vector, int size) {
  int min_index=0;
  int current_min=INT_MAX, i;

  for(i = 0; i < size; i++) {
    if( vector[i] < current_min ) {
      current_min = vector[i];
      min_index = i;
    }
  }

  return min_index;
}

int findMax(int *vector, int size) {
  int max_index=0;
  int current_max=INT_MIN, i;

  for(i = 0; i < size; i++) {
    if( vector[i] < current_max ) {
      current_max = vector[i];
      max_index = i;
    }
  }

  return max_index;
}

