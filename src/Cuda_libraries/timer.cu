#include "hip/hip_runtime.h"
#include <limits>
 
#include "timer.h"

Timer::Timer() {
  startTime = 0;
  endTime = 0;
  minTime = std::numeric_limits<double>::max();
  maxTime = std::numeric_limits<double>::min();
  totalTime = 0;
  counter = 0;
}

Timer::Timer(std::string whatAmI) {
  startTime = 0;
  endTime = 0;
  minTime = std::numeric_limits<double>::max();
  maxTime = std::numeric_limits<double>::min();
  totalTime = 0;
  counter = 0;
  description = whatAmI;
}
 
void Timer::startTimer() {
  counter++;
  startTime = MPI_Wtime();
}

void Timer::endTimer() {
  endTime = MPI_Wtime();
  double diff = endTime - startTime;
  totalTime += diff;
  if ( diff < minTime) minTime = diff;
  if ( diff > maxTime) maxTime = diff; 
}

void Timer::echoHeader(std::ofstream *log) {
  *log << "\t" << "Minimum Time (s)" << "\tMaximum Time (s)" << "\tAverage Time (s)" << std::endl; 
}
 
void Timer::echoTimer(std::ofstream *log) {
  *log << this->whatAmI() << "\t" << minTime << "\t" << maxTime << "\t" << totalTime/counter << std::endl;  
}

std::string Timer::whatAmI() {
  return description;
} 

/*Timer& Timer::operator=( const Timer &clock) {
  this->startTime = clock.startTime;
  this->endTime = clock.endTime;
  this->minTime = clock.minTime;
  this->maxTime = clock.maxTime;
  this->totalTime = clock.totalTime;
  this->counter = clock.counter;
  this->description = clock.description;
  return *this;
}
*/
