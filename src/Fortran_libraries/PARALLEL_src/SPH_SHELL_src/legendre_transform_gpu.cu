#include "legendre_transform_gpu.h"
#include <hip/hip_runtime.h>
#include <iostream>

__host__ void LegendreTransform::initialize_gpu()
{
  int device_count, device;
  hipGetDeviceCount(&device_count);
  hipGetDevice(&device);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  size_t devMemory = prop.totalGlobalMem;
  std::cout << "Device Memory = " << devMemory << std::endl;
  hipFree(0);
}

__host__ void LegendreTransform::finalize_gpu()
{
  hipFree(0);
}

extern "C" {
  void initialize_gpu_() {
    LegendreTransform x;
    x.initialize_gpu();
  }
}
